#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <iomanip>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <complex>

#include <hip/hip_complex.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <thrust/complex.h>
#include <thrust/for_each.h>
#include <thrust/transform.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>


#include "hip/hip_runtime_api.h"
#include "helper_string.h"

typedef thrust::complex<float> comThr;

__global__ void circshiftInTime(float* data, int n, int shift)
{
    int row = blockIdx.x; // row index
    int tid = threadIdx.x; // thread index within a block
    int num_threads = blockDim.x; // number of threads in a block

    // calculate the starting and ending indices of the portion of the row to be processed by this thread
    int start_idx = tid * (n / num_threads);
    int end_idx = (tid + 1) * (n / num_threads);
    if (tid == num_threads - 1) {
        end_idx = n; // handle the remainder in the last thread
    }

    // apply circshift to the portion of the row
    for (int i = start_idx; i < end_idx; i++) {
        int idx = row * n + i;
        int shifted_idx = (i + shift) % n + row * n;
        float tmp = data[idx];
        data[idx] = data[shifted_idx];
        data[shifted_idx] = tmp;
    }
}

template <typename T>
__global__ void circShiftKernel(T* d_in, T* d_out, int frag_len, int shift_num, int len)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < len)
    {
        //int base = static_cast<int>(tid / frag_len) * frag_len;
        //int offset = (tid % frag_len + shift_num) % frag_len;
        d_out[static_cast<int>(tid / frag_len) * frag_len + (tid % frag_len + shift_num) % frag_len] = d_in[tid];
    }
}

template <typename T>
void circshift(T* d_data, int frag_len, int shift, int len)
{
    T* d_data_temp = nullptr;
    checkCudaErrors(hipMalloc((void**)&d_data_temp, sizeof(T) * len));

    dim3 block(256);  // block size
    dim3 grid((len + block.x - 1) / block.x);  // grid size
    circShiftKernel << <grid, block >> > (d_data, d_data_temp, frag_len, shift, len);
    checkCudaErrors(hipDeviceSynchronize());

    //checkCudaErrors(hipFree(d_data));
    //d_data = d_data_temp;
    checkCudaErrors(hipMemcpy(d_data, d_data_temp, sizeof(T) * len, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipFree(d_data_temp));
}

void testCircshift();

void cufftTest();

void cuBlasTest();

void test();

template <typename T>
__global__ void swap_range(T* a, T* b, int len)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < len) {
        T c = a[tid]; a[tid] = b[tid]; b[tid] = c;
    }
}


int main()
{
    testCircshift();
}

void test()
{
    int len = 10;
    float* h_arr = new float[len];
    for (int i = 0; i < len; ++i) {
        h_arr[i] = static_cast<float>(i + 1);
    }

    float* d_arr = nullptr;
    checkCudaErrors(hipMalloc((void**)&d_arr, sizeof(float) * len));
    checkCudaErrors(hipMemcpy(d_arr, h_arr, sizeof(float) * len, hipMemcpyHostToDevice));

    swap_range<float> << <1, len >> > (d_arr, d_arr + len / 2, len / 2);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(h_arr, d_arr, sizeof(float) * len, hipMemcpyDeviceToHost));
    for (int i = 0; i < len; ++i) {
        std::cout << h_arr[i] << " ";
    }
}

void testCircshift()
{
    int shift = 3;
    int echo = 2;
    int range = 20;
    int data_num = echo * range;

    std::complex<float>* h_data = new std::complex<float>[data_num];
    for (int i = 0; i < data_num; ++i) {
        h_data[i] = std::complex<float>(static_cast<float>(i), static_cast<float>(i));
    }

    //for (int i = 0; i < data_num; ++i) {
    //    if (i % range == 0) {
    //        std::cout << "\n";
    //    }
    //    std::cout << h_data[i] << " ";
    //}

    hipComplex* d_data = nullptr;
    checkCudaErrors(hipMalloc((void**)&d_data, sizeof(hipComplex) * data_num));
    checkCudaErrors(hipMemcpy(d_data, h_data, sizeof(hipComplex) * data_num, hipMemcpyHostToDevice));

    circshift(d_data, range, shift, data_num);

    checkCudaErrors(hipMemcpy(h_data, d_data, sizeof(hipComplex) * data_num, hipMemcpyDeviceToHost));

    for (int i = 0; i < data_num; ++i) {
        if (i % range == 0) {
            std::cout << "\n";
        }
        std::cout << h_data[i] << " ";
    }

}

void cuBlasTest()
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);


    int len = 5;
    float* h_arr = new float[len];
    //for (int i = 0; i < len; ++i) {
    //    h_arr[i] = static_cast<float>(i + 1);
    //}
    h_arr[0] = static_cast<float>(0);
    h_arr[1] = static_cast<float>(1);
    h_arr[2] = static_cast<float>(2);
    h_arr[3] = static_cast<float>(6);
    h_arr[4] = static_cast<float>(3);


    float* d_arr = nullptr;

    float alpha = 5;
    int max_idx = 0;

    checkCudaErrors(hipMalloc((void**)&d_arr, sizeof(float) * len));
    checkCudaErrors(hipMemcpy(d_arr, h_arr, sizeof(float) * len, hipMemcpyHostToDevice));

    //checkCudaErrors(hipblasSscal(handle, len, &alpha, d_arr, 1));

    checkCudaErrors(hipblasIsamax(handle, len, d_arr, 1, &max_idx));
    --max_idx;
    
    //checkCudaErrors(hipMemcpy(h_arr, d_arr, sizeof(float) * len, hipMemcpyDeviceToHost));
    //for (int i = 0; i < len; ++i) {
    //    std::cout << h_arr[i] << " ";
    //}

    std::cout << max_idx;
    
}

void cufftTest() {
    int len = 256 * 4020;
    std::vector<std::complex<float>> h_data(len);

    for (int i = 0; i < len; ++i) {
        h_data[i].real(static_cast<float>(i + 1));
        h_data[i].imag(static_cast<float>(i + 2));
    }

    hipComplex* d_data = nullptr;

    hipMalloc((void**)&d_data, sizeof(hipComplex) * len);
    hipMemcpy(d_data, h_data.data(), sizeof(hipComplex) * len, hipMemcpyHostToDevice);
    thrust::device_ptr<comThr> thr_d_data = thrust::device_pointer_cast(reinterpret_cast<comThr*>(d_data));


    //for (int i = 0; i < len; ++i) {
    //    std::cout << thr_d_data[i] << " ";
    //} std::cout << "\n";

    
    hipfftHandle plan;
    hipfftPlan1d(&plan, len, HIPFFT_C2C, 1);

    auto tS = std::chrono::high_resolution_clock::now();
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
    auto tE = std::chrono::high_resolution_clock::now();
    std::cout << "[cuFFT] " << std::chrono::duration_cast<std::chrono::milliseconds>(tE - tS).count() << "ms\n";


    //for (int i = 0; i < len; ++i) {
    //    std::cout << thr_d_data[i] << " ";
    //} std::cout << "\n";

    hipfftDestroy(plan);

}
