#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <iomanip>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <complex>

#include <hip/hip_complex.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <thrust/complex.h>
#include <thrust/for_each.h>
#include <thrust/transform.h>
#include <thrust/execution_policy.h>

typedef thrust::complex<float> comThr;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

void cuBlasTest();

void originTest();

void cufftTest();

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    //hipComplex d_data = make_hipComplex(1.0f, 1.0f);
    hipComplex d_data{};
    d_data.x = 12.0f;
    d_data.y = 1.0f;

    hipComplex* d_ptr = &d_data;
    thrust::device_ptr<comThr> thr_d_ptr = thrust::device_pointer_cast(reinterpret_cast<comThr*>(d_ptr));
    std::cout << *thr_d_ptr;


    //hipMalloc((void**)&d_data, sizeof(hipComplex) * 2);
    //d_data[0].x = 

    //cufftTest();

    return 0;
}

void cufftTest() {
    int len = 5;
    std::vector<std::complex<float>> h_data(len);
    h_data[0].real(0.0f); h_data[0].imag(0.0f);
    h_data[1].real(1.0f); h_data[1].imag(0.0f);
    h_data[2].real(2.0f); h_data[2].imag(0.0f);
    h_data[3].real(3.0f); h_data[3].imag(0.0f);
    h_data[4].real(4.0f); h_data[4].imag(0.0f);
    //h_data[5].real(0.0f); h_data[5].imag(0.0f);
    //h_data[6].real(1.0f); h_data[6].imag(0.0f);
    //h_data[7].real(2.0f); h_data[7].imag(0.0f);
    //h_data[8].real(3.0f); h_data[8].imag(0.0f);
    //h_data[9].real(4.0f); h_data[9].imag(0.0f);

    hipComplex* d_data = nullptr;

    hipMalloc((void**)&d_data, sizeof(hipComplex) * len);
    hipMemcpy(d_data, h_data.data(), sizeof(hipComplex) * len, hipMemcpyHostToDevice);
    thrust::device_ptr<comThr> thr_d_data = thrust::device_pointer_cast(reinterpret_cast<comThr*>(d_data));


    for (int i = 0; i < len; ++i) {
        std::cout << thr_d_data[i] << " ";
        if (i == 4) std::cout << "\n";
    }
    std::cout << "\n";


    hipfftHandle plan;
    hipfftPlan1d(&plan, 5, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);

    for (int i = 0; i < len; ++i) {
        std::cout << thr_d_data[i] << " ";
        if (i == 4) std::cout << "\n";
    }
    std::cout << "\n";

    hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);
    thrust::transform(thrust::device, thr_d_data, thr_d_data + len, thr_d_data, [=]__host__ __device__(const comThr & x) { return x / static_cast<float>(len); });

    for (int i = 0; i < len; ++i) {
        std::cout << thr_d_data[i] << " ";
        if (i == 4) std::cout << "\n";
    }
    std::cout << "\n";

}


void cuBlasTest() {
    std::vector<float> echo({ 1,2,1 });
    std::vector<float> range({ 1,2,3,4,5 });

    int echo_num = echo.size();
    int range_num = range.size();
    int data_num = range_num * echo_num;

    float* d_echo = nullptr;
    hipMalloc((void**)&d_echo, sizeof(float) * echo_num);
    hipMemcpy(d_echo, echo.data(), sizeof(float) * echo_num, hipMemcpyHostToDevice);  // data (host -> device)


    thrust::device_ptr<float> thr_d_echo(d_echo);
    for (int i = 0; i < echo_num; ++i) {
        std::cout << thr_d_echo[i] << " ";
    }
    std::cout << "\n";

    float* d_range = nullptr;
    hipMalloc((void**)&d_range, sizeof(float) * range_num);
    hipMemcpy(d_range, range.data(), sizeof(float) * range_num, hipMemcpyHostToDevice);  // data (host -> device)

    thrust::device_ptr<float> thr_d_range(d_range);
    for (int i = 0; i < range_num; ++i) {
        std::cout << thr_d_range[i] << " ";
    }
    std::cout << "\n";

    float* res = nullptr;
    hipMalloc((void**)&res, sizeof(float) * data_num);
    hipMemset(res, 0.0f, sizeof(float) * data_num);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float a = 1;
    hipblasSger(handle, range_num, echo_num, &a, d_range, 1, d_echo, 1, res, range_num);

    thrust::device_ptr<float> thr_d_res(res);
    for (int i = 0; i < data_num; ++i) {
        std::cout << thr_d_res[i] << " ";
    }
    std::cout << "\n";

    hipblasDestroy(handle);

    float* h_data = new float[data_num];
    hipMemcpy(h_data, res, sizeof(float) * data_num, hipMemcpyDeviceToHost);

    for (int i = 0; i < data_num; ++i) {
        std::cout << h_data[i] << " ";
    }
}


void originTest() {
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return ;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return ;
    }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
