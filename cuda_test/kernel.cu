#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <iomanip>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <complex>
#include <type_traits>

#include <hip/hip_complex.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <thrust/complex.h>
#include <thrust/for_each.h>
#include <thrust/transform.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>


#include "hip/hip_runtime_api.h"
#include "helper_string.h"

typedef thrust::complex<float> comThr;
constexpr auto PI_h = 3.14159265358979f;

//void cufftTest()
//{
//    // generate data of size echo * range, each element is a float number, pointed by h_data
//    int echo = 4;
//    int range = 6;
//    int data_num = echo * range;
//    float* h_data = new float[echo * range];
//    for (int i = 0; i < data_num; ++i) {
//        h_data[i] = static_cast<float>(i);
//    }
//
//    // print h_data
//    std::cout << "h_data: \n";
//    for (int i = 0; i < data_num; ++i) {
//        if (i % range == 0) {
//            std::cout << "\n";
//        }
//        std::cout << h_data[i] << " ";
//    }
//
//    // copy data from host to device
//    float* d_data = nullptr;
//    checkCudaErrors(hipMalloc((void**)&d_data, sizeof(float) * data_num));
//    checkCudaErrors(hipMemcpy(d_data, h_data, sizeof(float) * data_num, hipMemcpyHostToDevice));
//
//    hipComplex* d_data_c = nullptr;
//    checkCudaErrors(hipMalloc((void**)&d_data_c, sizeof(hipComplex) * echo * (range / 2 + 1)));
//
//    // create cufft plan
//    hipfftHandle plan;
//    checkCudaErrors(hipfftPlan1d(&plan, range, HIPFFT_R2C, echo));
//    // execute cufft plan
//    checkCudaErrors(hipfftExecR2C(plan, d_data, d_data_c));
//    // copy data from device to host
//    std::complex<float>* h_data_c = new std::complex<float>[echo * (range / 2 + 1)];
//    checkCudaErrors(hipMemcpy(h_data_c, d_data_c, sizeof(std::complex<float>) * echo * (range / 2 + 1), hipMemcpyDeviceToHost));
//    // print data
//    std::cout << "\n\nh_data_c: \n";
//    for (int i = 0; i < echo * (range / 2 + 1); ++i) {
//        if (i % (range / 2 + 1) == 0) {
//            std::cout << "\n";
//        }
//        std::cout << h_data_c[i] << " ";
//    }
//
//    // ifft
//    // culabs handle
//    hipblasHandle_t handle;
//    checkCudaErrors(hipblasCreate(&handle));
//
//    // create cufft plan
//    hipfftHandle plan2;
//    checkCudaErrors(hipfftPlan1d(&plan2, range, HIPFFT_C2R, echo));
//    // execute cufft plan
//    checkCudaErrors(hipfftExecC2R(plan2, d_data_c, d_data));
//    // d_data divided by range
//    float scal_ifft = 1 / static_cast<float>(range);
//    checkCudaErrors(hipblasSscal(handle, data_num, &scal_ifft, d_data, 1));
//    // copy data from device to host
//    checkCudaErrors(hipMemcpy(h_data, d_data, sizeof(float) * data_num, hipMemcpyDeviceToHost));
//    // print data
//    std::cout << "\n\nh_data: \n";
//    for (int i = 0; i < data_num; ++i) {
//        if (i % range == 0) {
//            std::cout << "\n";
//        }
//        std::cout << h_data[i] << " ";
//    }
//}


// copy device data back to host and display
void dDataDisp(float* d_data, int rows, int cols)
{
    float* h_data = new float[rows * cols];
    checkCudaErrors(hipMemcpy(h_data, d_data, sizeof(float) * rows * cols, hipMemcpyDeviceToHost));
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << h_data[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
    delete[] h_data;
}


void dDataDisp(hipComplex* d_data, int rows, int cols)
{
    std::complex<float>* h_data = new std::complex<float>[rows * cols];
    checkCudaErrors(hipMemcpy(h_data, d_data, sizeof(std::complex<float>) * rows * cols, hipMemcpyDeviceToHost));
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << h_data[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
    delete[] h_data;
}


int main(int argc, char** argv)
{
    hipComplex* d_data = new hipComplex[3];
    d_data[0] = make_hipComplex(0.0f, 0.0f);
    d_data[1] = make_hipComplex(1.0f, 0.0f);
    d_data[2] = make_hipComplex(2.0f, 0.0f);

    hipComplex* d_data_tmp = new hipComplex[1];

    checkCudaErrors(hipMemcpy(d_data_tmp, d_data, sizeof(hipComplex) * 1, hipMemcpyHostToHost));
    std::cout << d_data_tmp->x << " " << d_data_tmp->y << std::endl;

    checkCudaErrors(hipMemcpy(d_data_tmp, d_data + 1, sizeof(hipComplex) * 1, hipMemcpyHostToHost));
    std::cout << d_data_tmp->x << " " << d_data_tmp->y << std::endl;

    checkCudaErrors(hipMemcpy(d_data_tmp, d_data + 2, sizeof(hipComplex) * 1, hipMemcpyHostToHost));
    std::cout << d_data_tmp->x << " " << d_data_tmp->y << std::endl;

 //   int echo_num = 2;
 //   int range_num = 6;
 //   int data_num = echo_num * range_num;
 //   float scale_ifft_range = 1.0f / range_num;
 //   float scale_ifft_echo = 1.0f / echo_num;
 //   dim3 block(256);

 //   hipblasHandle_t handle;
 //   checkCudaErrors(hipblasCreate(&handle));

 //   hipfftHandle plan_all_echo_c2c_cut;
	//checkCudaErrors(hipfftPlan1d(&plan_all_echo_c2c_cut, range_num, HIPFFT_C2C, echo_num));

 //   hipfftHandle plan_all_range_c2c;
 //   hipfftHandle plan_all_range_c2c_czt;
 //   int batch = range_num;
 //   int rank = 1;
 //   int n[] = { echo_num };
 //   int inembed[] = { echo_num };
 //   int onembed[] = { echo_num };
 //   int istride = range_num;
 //   int ostride = range_num;
 //   int idist = 1;
 //   int odist = 1;
 //   checkCudaErrors(hipfftPlanMany(&plan_all_range_c2c, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

 //   int fft_len = nextPow2(2 * echo_num - 1);
 //   n[0] = fft_len;
 //   inembed[0] = fft_len;
 //   onembed[0] = fft_len;
 //   checkCudaErrors(hipfftPlanMany(&plan_all_range_c2c_czt, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

 //   // initializing all element of d_data
 //   std::complex<float>* h_data = new std::complex<float>[data_num];
 //   for (int i = 0; i < data_num; ++i) {
 //       h_data[i] = std::complex<float>(static_cast<float>(i + 1), 0.0f);
 //   }
 //   hipComplex* d_data = nullptr;
 //   checkCudaErrors(hipMalloc((void**)&d_data, sizeof(hipComplex) * data_num));
 //   checkCudaErrors(hipMemcpy(d_data, h_data, sizeof(hipComplex) * data_num, hipMemcpyHostToDevice));
 //   hipComplex* d_st = nullptr;
 //   checkCudaErrors(hipMalloc((void**)&d_st, sizeof(hipComplex) * data_num));
 //   checkCudaErrors(hipMemcpy(d_st, d_data, sizeof(hipComplex) * data_num, hipMemcpyDeviceToDevice));
 //   // print d_data
 //   std::cout << "d_data:" << std::endl;
 //   dDataDisp(d_data, echo_num, range_num);

 //   // ifftshift
 //   ifftshiftRows << <dim3(((range_num / 2) + block.x - 1) / block.x, echo_num), block >> > (d_st, range_num);
 //   checkCudaErrors(hipDeviceSynchronize());
 //   // ifft
 //   checkCudaErrors(hipfftExecC2C(plan_all_echo_c2c_cut, d_st, d_st, HIPFFT_BACKWARD));
 //   checkCudaErrors(hipblasCsscal(handle, data_num, &scale_ifft_range, d_st, 1));
 //   // print d_data
 //   std::cout << "d_st:" << std::endl;
 //   dDataDisp(d_st, echo_num, range_num);

 //   // * CZT
 //   // calculating w and a vector for each range
 //   hipComplex* d_w = nullptr;
 //   checkCudaErrors(hipMalloc((void**)&d_w, sizeof(hipComplex) * range_num));
 //   hipComplex* d_a = nullptr;
 //   checkCudaErrors(hipMalloc((void**)&d_a, sizeof(hipComplex) * range_num));

 //   float constant = 0.0601f;
 //   float posa = 2.2579e-04f;
 //   getWandA << <(2 * range_num + block.x - 1) / block.x, block >> > (d_w, d_a, echo_num, range_num, constant, posa);
 //   checkCudaErrors(hipDeviceSynchronize());
 //   std::cout << "d_w:" << std::endl;
 //   dDataDisp(d_w, 1, range_num);
 //   std::cout << "d_a:" << std::endl;
 //   dDataDisp(d_a, 1, range_num);

 //   // CZT
 //   // nfft = 2^nextpow2(m+k-1);
 //   float scale_ifft = 1.0f / fft_len;
 //   int data_num_fft = fft_len * range_num;
 //   int ww_len = 2 * echo_num - 1;  // ww length for each range: 2 * echo_num - 1
 //   int y_len = echo_num;  // y length for each range: echo_num

 //   hipComplex* d_ww = nullptr;
 //   checkCudaErrors(hipMalloc((void**)&d_ww, sizeof(hipComplex) * data_num_fft));
 //   thrust::device_ptr<comThr> thr_ww = thrust::device_pointer_cast(reinterpret_cast<comThr*>(d_ww));
 //   genWW << <dim3(range_num, (fft_len + block.x - 1) / block.x), block >> > (d_ww, d_w, echo_num, range_num, ww_len, fft_len);
 //   checkCudaErrors(hipDeviceSynchronize());

 //   hipComplex* d_y = nullptr;
 //   checkCudaErrors(hipMalloc((void**)&d_y, sizeof(hipComplex) * data_num_fft));
 //   gety << <dim3(range_num, (fft_len + block.x - 1) / block.x), block >> > (d_y, d_a, d_ww, d_st, echo_num, range_num, y_len, fft_len);
 //   checkCudaErrors(hipDeviceSynchronize());

 //   // fft
 //   checkCudaErrors(hipfftExecC2C(plan_all_range_c2c_czt, d_y, d_y, HIPFFT_FORWARD));

 //   hipComplex* d_ww_ = nullptr;
 //   checkCudaErrors(hipMalloc((void**)&d_ww_, sizeof(hipComplex) * data_num_fft));
 //   thrust::device_ptr<comThr> thr_ww_ = thrust::device_pointer_cast(reinterpret_cast<comThr*>(d_ww_));
 //   thrust::transform(thrust::device, thr_ww, thr_ww + data_num_fft - (fft_len - ww_len) * range_num, thr_ww_, \
 //       []__host__ __device__(const comThr & x) { return thrust::pow(x, -1); });

 //   checkCudaErrors(hipfftExecC2C(plan_all_range_c2c_czt, d_ww_, d_ww_, HIPFFT_FORWARD));

 //   elementwiseMultiply << <(data_num_fft + block.x - 1) / block.x, block >> > (d_y, d_ww_, d_y, data_num_fft);
 //   checkCudaErrors(hipDeviceSynchronize());

 //   // ifft
 //   checkCudaErrors(hipfftExecC2C(plan_all_range_c2c_czt, d_y, d_y, HIPFFT_BACKWARD));
 //   checkCudaErrors(hipblasCsscal(handle, data_num_fft, &scale_ifft, d_y, 1));
 //   std::cout << "d_y(ifft):" << std::endl;
 //   dDataDisp(d_y, fft_len, range_num);

 //   hipComplex* d_czt = d_st;
 //   getCZTOut << <dim3(range_num, (echo_num + block.x - 1) / block.x), block >> > (d_czt, d_y, d_ww, echo_num);
 //   checkCudaErrors(hipDeviceSynchronize());
 //   std::cout << "d_czt:" << std::endl;
 //   dDataDisp(d_czt, echo_num, range_num);
}
