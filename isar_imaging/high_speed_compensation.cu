#include "hip/hip_runtime.h"
﻿#include "high_speed_compensation.cuh"

void highSpeedCompensation(hipComplex* d_data, int Fs, long long band_width, float Tp, float* d_velocity, int echo_num, int range_num, hipblasHandle_t handle)
{
	int data_num = echo_num * range_num;

	dim3 block(256);  // block size
	dim3 grid((data_num + block.x - 1) / block.x);  // grid size
	dim3 grid_range((range_num + block.x - 1) / block.x);  // grid size

	// fast time vector
	float* d_tk_2 = nullptr;  // tk_2 = ([0:N-1]/fs).^2
	checkCudaErrors(hipMalloc((void**)&d_tk_2, sizeof(float)* range_num));
	genTk2Vec << <grid_range, block >> > (d_tk_2, static_cast<float>(Fs), range_num);
	checkCudaErrors(hipDeviceSynchronize());

	// coef = - 4 * pi * K / c
	float chirp_rate = -static_cast<float>(band_width) / Tp;  // extra minus symbol for velocity (depending on different radar signal)
	float coefficient = 4.0f * PI_h * chirp_rate / lightSpeed_h;

	// phase = coef * v * tk.^2
	float* d_phase = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_phase, sizeof(float) * data_num));  // new mallocated space are set to zero
	checkCudaErrors(hipblasSger(handle, range_num, echo_num, &coefficient, d_tk_2, 1, d_velocity, 1, d_phase, range_num));

	// phi = exp(1j*phase)
	hipComplex* d_phi = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_phi, sizeof(hipComplex)* data_num));
	expJ << <grid, block >> > (d_phase, d_phi, data_num);
	checkCudaErrors(hipDeviceSynchronize());

	// compensation
	elementwiseMultiply << <grid, block >> > (d_data, d_phi, d_data, data_num);
	checkCudaErrors(hipDeviceSynchronize());

	// free gpu mallocated space
	checkCudaErrors(hipFree(d_phase));
	checkCudaErrors(hipFree(d_phi));
	checkCudaErrors(hipFree(d_tk_2));
}


__global__ void genTk2Vec(float* tk2, float Fs, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len) {
		tk2[tid] = (static_cast<float>(tid) / Fs) * (static_cast<float>(tid) / Fs);
	}
}