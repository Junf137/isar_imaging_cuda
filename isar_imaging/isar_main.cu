#include "hip/hip_runtime.h"
﻿#include "isar_main.cuh"

int ISAR_RD_Imaging_Main_Ku(RadarParameters& paras, const int& datastyle, const vec1D_COM_FLOAT& dataW, const vec2D_FLOAT& dataNOut, const int& optionAligment, const int& optionAPhase, const bool& ifHPC, const bool& ifMTRC) {
	/******************************
	* Init GPU Device
	******************************/
	int devID = 0;  // pick the device with highest Gflops/s. (single GPU mode)
	checkCudaErrors(hipSetDevice(devID));

	// * CUDA Compability Information
	//int major = 0, minor = 0;
	//checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, devID));
	//checkCudaErrors(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, devID));
	//printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, _ConvertSMVer2ArchName(major, minor), major, minor);

	
	/******************************
	* GPU Memory Initialization
	******************************/
	std::cout << "---* Starting GPU Memory Initialization *---\n";
	auto tStart_InitGPU = std::chrono::high_resolution_clock::now();

	const std::complex<float>* h_data = dataW.data();

	int data_num = paras.echo_num * paras.range_num;

	// * Overall cuBlas handle
	hipblasHandle_t handle;
	checkCudaErrors(hipblasCreate(&handle));

	// * Overall cuFFT plan
	hipfftHandle plan_all_echo_c2c;
	checkCudaErrors(hipfftPlan1d(&plan_all_echo_c2c, paras.range_num, HIPFFT_C2C, paras.echo_num));
	hipfftHandle plan_one_echo_c2c;
	checkCudaErrors(hipfftPlan1d(&plan_one_echo_c2c, paras.range_num, HIPFFT_C2C, 1));
	hipfftHandle plan_one_echo_r2c;  // implicitly forward
	checkCudaErrors(hipfftPlan1d(&plan_one_echo_r2c, paras.range_num, HIPFFT_R2C, 1));
	hipfftHandle plan_one_echo_c2r;  // implicitly inverse
	checkCudaErrors(hipfftPlan1d(&plan_one_echo_c2r, paras.range_num, HIPFFT_C2R, 1));

	// * Overall kernal function configuration
	dim3 block(256);  // block size
	dim3 grid((data_num + block.x - 1) / block.x);  // grid size
	dim3 grid_range((paras.range_num + block.x - 1) / block.x);

	// * GPU memory mallocation
	hipComplex* d_data = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_data, sizeof(hipComplex) * data_num));
	checkCudaErrors(hipMemcpy(d_data, h_data, sizeof(hipComplex) * data_num, hipMemcpyHostToDevice));  // data (host -> device)
	thrust::device_ptr<comThr> thr_d_data = thrust::device_pointer_cast(reinterpret_cast<comThr*>(d_data));

	auto tEnd_InitGPU = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_InitGPU - tStart_InitGPU).count() << "ms\n";
	std::cout << "---* GPU Memory Initialization Over *---\n";
	std::cout << "************************************\n\n";


#ifdef DATA_WRITE_BACK
	ioOperation::dataWriteBack(std::string(DIR_PATH) + "dataW.dat", d_data, data_num);
#endif // DATA_WRITE_BACK


	/******************************
	* HPC
	******************************/
	if (ifHPC == true) {
		std::cout << "---* Starting HPC *---\n";

		// * Retrieving Velocity Data
		float* h_velocity = new float[paras.echo_num];
		std::transform(dataNOut.cbegin(), dataNOut.cend(), h_velocity, [](const std::vector<float>& v) {return v[1]; });

		float* d_velocity = nullptr;
		checkCudaErrors(hipMalloc((void**)&d_velocity, sizeof(float) * paras.echo_num));
		checkCudaErrors(hipMemcpy(d_velocity, h_velocity, sizeof(float) * paras.echo_num, hipMemcpyHostToDevice));

		auto tStart_HPC = std::chrono::high_resolution_clock::now();

		// * Starting HPC
		highSpeedCompensation(d_data, paras.Fs, paras.band_width, paras.Tp, d_velocity, paras.echo_num, paras.range_num, handle);
		
		auto tEnd_HPC = std::chrono::high_resolution_clock::now();

		delete[] h_velocity;
		h_velocity = nullptr;
		checkCudaErrors(hipFree(d_velocity));
		d_velocity = nullptr;

		std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_HPC - tStart_HPC).count() << "ms\n";
		std::cout << "---* HPC Over *---\n";
		std::cout << "************************************\n\n";
	}


#ifdef DATA_WRITE_BACK
	ioOperation::dataWriteBack(std::string(DIR_PATH) + "hpc.dat", d_data, data_num);
#endif // DATA_WRITE_BACK


	/******************
	 * HRRP
	 ******************/
	std::cout << "---* Starting Get HRRP *---\n";

	// * Generate Hamming Window
	float* hamming = nullptr;
	checkCudaErrors(hipMalloc((void**)&hamming, sizeof(float) * paras.range_num));
	genHammingVec << <grid_range, block >> > (hamming, paras.range_num);
	checkCudaErrors(hipDeviceSynchronize());

	// * HRRP - High Resolution Range Profile.
	hipComplex* d_hrrp = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_hrrp, sizeof(hipComplex) * data_num));

	auto tStart_HRRP = std::chrono::high_resolution_clock::now();

	// d_hrrp = fftshift(fft(hamming ,* d_data))
	getHRRP(d_hrrp, d_data, paras.echo_num, paras.range_num, hamming, plan_all_echo_c2c);

	auto tEnd_HRRP = std::chrono::high_resolution_clock::now();

	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_HRRP - tStart_HRRP).count() << "ms\n";
	std::cout << "---* Get HRRP Over *---\n";
	std::cout << "************************************\n\n";


	/******************
	 * Range Alignment and HRRP Centering
	 ******************/
	std::cout << "---* Starting Range alignment *---\n";
	auto tStart_RA = std::chrono::high_resolution_clock::now();

	// * Range Alignment
	rangeAlignment(d_data, hamming, paras, handle, plan_one_echo_c2c, plan_one_echo_r2c, plan_one_echo_c2r);

	auto tEnd_RA_1 = std::chrono::high_resolution_clock::now();

	// * Centering HRRP
	unsigned int inter_length = 30;
	HRRPCenter(d_data, paras, inter_length, handle, plan_all_echo_c2c);

	auto tEnd_RA_2 = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption(range alignment)] " << std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_RA_1 - tStart_RA).count() << "ms\n";
	std::cout << "[Time consumption(centering HRRP)] " << std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_RA_2 - tEnd_RA_1).count() << "ms\n";
	std::cout << "---* Range alignment Over *---\n";
	std::cout << "************************************\n\n";


	/******************
	* Cut range profiles
	******************/
	std::cout << "---* Starting Cut range profiles *---\n";
	auto tStart_cut = std::chrono::high_resolution_clock::now();

	const int range_length = 512;

	hipComplex* d_data_cut = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_data_cut, sizeof(hipComplex)* range_length* paras.echo_num));

	// YSQ 改为求最大值点，确定开始距离单元
	float* range_abs = nullptr;  // 求幅度
	checkCudaErrors(hipMalloc((void**)&range_abs, sizeof(float)* paras.range_num));
	thrust::device_ptr<float> thr_range_abs(range_abs);
	thrust::transform(thrust::device, thr_d_data, thr_d_data + paras.range_num, thr_range_abs,
		[]__host__ __device__(const thrust::complex<float>& x) { return thrust::abs(x); });
	thrust::device_ptr<float> min_ptr = thrust::max_element(thr_range_abs, thr_range_abs + paras.range_num);

	int mPos = static_cast<int>(&min_ptr[0] - &thr_range_abs[0]);
	paras.Pos = mPos;  // modify paras value

	cutRangeProfile(d_data, d_data_cut, range_length, paras);
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(range_abs));
	auto tEnd_cut = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_cut - tStart_cut).count() << "ms\n";
	std::cout << "---* Cut range profiles Over *---\n";
	std::cout << "************************************\n\n";

	checkCudaErrors(hipFree(d_data));  // use d_data_cut instead
	paras.range_num = range_length;  // modify paras value
	data_num = paras.echo_num * paras.range_num;


	/**********************
	 * Phase Compensation
	 * 多普勒跟踪 -> 距离向空变的相位补偿 -> 快速最小熵 (Doppler_Tracking -> RangeVariantPhaseComp -> Fast_Entropy)
	 **********************/
	float* h_azimuth = new float[paras.echo_num];
	float* h_pitch = new float[paras.echo_num];
	std::transform(dataNOut.cbegin(), dataNOut.cend(), h_azimuth, [](std::vector<float> v) {return v[2]; });  // make zip with range and velocity data
	std::transform(dataNOut.cbegin(), dataNOut.cend(), h_pitch, [](std::vector<float> v) {return v[3]; });

	// Doppler centriod tracing
	std::cout << "---* Starting Doppler centriod tracing *---\n";
	auto tStart_droptrace = std::chrono::high_resolution_clock::now();

	Doppler_Tracking(d_data_cut, paras);
	checkCudaErrors(hipDeviceSynchronize());
	
	auto tEnd_droptrace = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_droptrace - tStart_droptrace).count() << "ms\n";
	std::cout << "---* Doppler centriod tracing Over *---\n";
	std::cout << "************************************\n\n";

	// range variant phase compensation
	std::cout << "---* Starting range variant phase compensation *---\n";
	auto tStart_ran = std::chrono::high_resolution_clock::now();

	RangeVariantPhaseComp(d_data_cut, paras, h_azimuth, h_pitch);

	checkCudaErrors(hipDeviceSynchronize());
	auto tEnd_ran = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_ran - tStart_ran).count() << "ms\n";
	std::cout << "---* range variant phase compensation Over *---\n";
	std::cout << "************************************\n\n";

	// 
	Fast_Entropy(d_data_cut, paras);


	ioOperation::dataWriteBack(std::string(DIR_PATH) + "isar_image.dat", d_data_cut, data_num);


	/**********************
	* Free Allocated Memory & Destory Pointer
	**********************/
	delete[] h_azimuth;
	h_azimuth = nullptr;
	delete[] h_pitch;
	h_pitch = nullptr;
	h_data = nullptr;  // h_data cannot be deleted (pointer resides in vector)

	checkCudaErrors(hipFree(hamming));
	hamming = nullptr;
	checkCudaErrors(hipFree(d_hrrp));
	d_hrrp = nullptr;
	
	checkCudaErrors(hipblasDestroy(handle));
	
	checkCudaErrors(hipfftDestroy(plan_all_echo_c2c));
	checkCudaErrors(hipfftDestroy(plan_one_echo_c2c));
	return EXIT_SUCCESS;
}