﻿#include "isar_main.cuh"

int ISAR_RD_Imaging_Main_Ku(RadarParameters& paras, const int& datastyle, const vec1D_COM_FLOAT& dataW, const vec2D_FLOAT& dataNOut, const int& optionAligment, const int& optionAPhase, const bool& ifHPC, const bool& ifMTRC) {
	/******************************
	* Init GPU Device
	******************************/
	int devID = 0;  // pick the device with highest Gflops/s. (single GPU mode)
	checkCudaErrors(hipSetDevice(devID));

	// * CUDA Compability Information
	//int major = 0, minor = 0;
	//checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, devID));
	//checkCudaErrors(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, devID));
	//printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, _ConvertSMVer2ArchName(major, minor), major, minor);

	
	/******************************
	* GPU Memory Initialization
	******************************/
	std::cout << "---* Starting GPU Memory Initialization *---\n";
	auto tStart_InitGPU = std::chrono::high_resolution_clock::now();

	const std::complex<float>* h_data = dataW.data();

	int data_num = paras.echo_num * paras.range_num;

	// * Overall cuBlas handle
	hipblasHandle_t handle;
	checkCudaErrors(hipblasCreate(&handle));

	// * Overall cuFFT plan
	hipfftHandle plan_all_echo_c2c;
	checkCudaErrors(hipfftPlan1d(&plan_all_echo_c2c, paras.range_num, HIPFFT_C2C, paras.echo_num));
	hipfftHandle plan_one_echo_c2c;
	checkCudaErrors(hipfftPlan1d(&plan_one_echo_c2c, paras.range_num, HIPFFT_C2C, 1));
	hipfftHandle plan_one_echo_r2c;  // implicitly forward
	checkCudaErrors(hipfftPlan1d(&plan_one_echo_r2c, paras.range_num, HIPFFT_R2C, 1));
	hipfftHandle plan_one_echo_c2r;  // implicitly inverse
	checkCudaErrors(hipfftPlan1d(&plan_one_echo_c2r, paras.range_num, HIPFFT_C2R, 1));

	// * Overall kernal function configuration
	dim3 block(256);  // block size
	dim3 grid((data_num + block.x - 1) / block.x);  // grid size
	dim3 grid_range((paras.range_num + block.x - 1) / block.x);

	// * GPU memory mallocation
	hipComplex* d_data = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_data, sizeof(hipComplex) * data_num));
	checkCudaErrors(hipMemcpy(d_data, h_data, sizeof(hipComplex) * data_num, hipMemcpyHostToDevice));  // data (host -> device)

	auto tEnd_InitGPU = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_InitGPU - tStart_InitGPU).count() << "ms\n";
	std::cout << "---* GPU Memory Initialization Over *---\n";
	std::cout << "************************************\n\n";


#ifdef DATA_WRITE_BACK
	ioOperation::dataWriteBack(std::string(DIR_PATH) + "dataW.dat", d_data, data_num);
#endif // DATA_WRITE_BACK


	/******************************
	* HPC
	******************************/
	if (ifHPC == true) {
		std::cout << "---* Starting HPC *---\n";

		// * Retrieving Velocity Data
		float* h_velocity = new float[paras.echo_num];
		std::transform(dataNOut.cbegin(), dataNOut.cend(), h_velocity, [](const std::vector<float>& v) {return v[1]; });

		float* d_velocity = nullptr;
		checkCudaErrors(hipMalloc((void**)&d_velocity, sizeof(float) * paras.echo_num));
		checkCudaErrors(hipMemcpy(d_velocity, h_velocity, sizeof(float) * paras.echo_num, hipMemcpyHostToDevice));

		auto tStart_HPC = std::chrono::high_resolution_clock::now();

		// * Starting HPC
		highSpeedCompensation(d_data, paras.Fs, paras.band_width, paras.Tp, d_velocity, paras.echo_num, paras.range_num, handle);
		
		auto tEnd_HPC = std::chrono::high_resolution_clock::now();

		delete[] h_velocity;
		h_velocity = nullptr;
		checkCudaErrors(hipFree(d_velocity));
		d_velocity = nullptr;

		std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::microseconds>(tEnd_HPC - tStart_HPC).count() << "us\n";
		std::cout << "---* HPC Over *---\n";
		std::cout << "************************************\n\n";
	}


#ifdef DATA_WRITE_BACK
	ioOperation::dataWriteBack(std::string(DIR_PATH) + "hpc.dat", d_data, data_num);
#endif // DATA_WRITE_BACK


	/******************
	 * HRRP
	 ******************/
	std::cout << "---* Starting Get HRRP *---\n";

	// * Generate Hamming Window
	float* hamming = nullptr;
	checkCudaErrors(hipMalloc((void**)&hamming, sizeof(float) * paras.range_num));
	genHammingVec << <grid_range, block >> > (hamming, paras.range_num);
	checkCudaErrors(hipDeviceSynchronize());

	// * HRRP - High Resolution Range Profile.
	hipComplex* d_hrrp = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_hrrp, sizeof(hipComplex) * data_num));

	auto tStart_HRRP = std::chrono::high_resolution_clock::now();

	// d_hrrp = fftshift(fft(hamming ,* d_data))
	getHRRP(d_hrrp, d_data, paras.echo_num, paras.range_num, hamming, plan_all_echo_c2c);

	auto tEnd_HRRP = std::chrono::high_resolution_clock::now();

	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::microseconds>(tEnd_HRRP - tStart_HRRP).count() << "us\n";
	std::cout << "---* Get HRRP Over *---\n";
	std::cout << "************************************\n\n";


	/******************
	 * Range Alignment and HRRP Centering
	 ******************/
	std::cout << "---* Starting Range alignment *---\n";
	auto tStart_RA = std::chrono::high_resolution_clock::now();

	// * Range Alignment
	rangeAlignment(d_data, hamming, paras, handle, plan_one_echo_c2c, plan_one_echo_r2c, plan_one_echo_c2r);

	auto tEnd_RA_1 = std::chrono::high_resolution_clock::now();

	// * Centering HRRP
	unsigned int inter_length = 30;
	HRRPCenter(d_data, paras, inter_length, handle, plan_all_echo_c2c);

	auto tEnd_RA_2 = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption(range alignment)] " << std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_RA_1 - tStart_RA).count() << "ms\n";
	std::cout << "[Time consumption(centering HRRP)] " << std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_RA_2 - tEnd_RA_1).count() << "ms\n";
	std::cout << "---* Range alignment Over *---\n";
	std::cout << "************************************\n\n";


	/******************
	* Cut range profiles
	******************/
	std::cout << "---* Starting Cut range profiles *---\n";
	auto tStart_cut = std::chrono::high_resolution_clock::now();

	int range_num_cut = 512;

	cutRangeProfile(d_data, paras, range_num_cut, handle);
	data_num = paras.echo_num * paras.range_num;

	auto tEnd_cut = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::microseconds>(tEnd_cut - tStart_cut).count() << "us\n";
	std::cout << "---* Cut range profiles Over *---\n";
	std::cout << "************************************\n\n";


	/**********************
	 * Phase Compensation
	 * 多普勒跟踪 -> 距离向空变的相位补偿 -> 快速最小熵 (Doppler_Tracking -> RangeVariantPhaseComp -> Fast_Entropy)
	 **********************/
	std::cout << "---* Starting Phase Compensation *---\n";

	// * Retrieving Azimith and Pitch Data
	float* h_azimuth = new float[paras.echo_num];
	float* h_pitch = new float[paras.echo_num];
	std::transform(dataNOut.cbegin(), dataNOut.cend(), h_azimuth, [](std::vector<float> v) {return v[2]; });
	std::transform(dataNOut.cbegin(), dataNOut.cend(), h_pitch, [](std::vector<float> v) {return v[3]; });

	auto tPC_1 = std::chrono::high_resolution_clock::now();

	// * Doppler Tracking
	dopplerTracking(d_data, paras.echo_num, paras.range_num);
	
	auto tPC_2 = std::chrono::high_resolution_clock::now();

	// * Range Variant Phase Compensation
	rangeVariantPhaseComp(d_data, paras, h_azimuth, h_pitch, handle);

	auto tPC_3 = std::chrono::high_resolution_clock::now();

	// * Fast Entropy
	fastEntropy(d_data, paras.echo_num, paras.range_num, handle);

	auto tPC_4 = std::chrono::high_resolution_clock::now();

	delete[] h_azimuth;
	h_azimuth = nullptr;
	delete[] h_pitch;
	h_pitch = nullptr;

	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::microseconds>(tPC_2 - tPC_1).count() << "us\n";
	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::microseconds>(tPC_3 - tPC_2).count() << "us\n";
	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::microseconds>(tPC_4 - tPC_3).count() << "us\n";
	std::cout << "---* Phase Compensation Over *---\n";
	std::cout << "************************************\n\n";


	/**********************
	* Final Data Write Back
	**********************/
	ioOperation::dataWriteBack(std::string(DIR_PATH) + "isar_image.dat", d_data, data_num);


	/**********************
	* Free Allocated Memory & Destory Pointer
	**********************/
	checkCudaErrors(hipFree(d_data));
	checkCudaErrors(hipFree(hamming));
	checkCudaErrors(hipFree(d_hrrp));
	
	checkCudaErrors(hipblasDestroy(handle));
	
	checkCudaErrors(hipfftDestroy(plan_all_echo_c2c));
	checkCudaErrors(hipfftDestroy(plan_one_echo_c2c));
	return EXIT_SUCCESS;
}