﻿#include "isar_main.cuh"

int ISAR_RD_Imaging_Main_Ku(RadarParameters& paras, const int& data_style, const vec1D_COM_FLOAT& dataW, const vec2D_FLOAT& dataNOut, const int& option_alignment, const int& option_phase, const bool& if_hpc, const bool& if_mtrc)
{
	/******************************
	* Init GPU Device
	******************************/
	int devID = 0;  // pick the device with highest Gflops/s. (single GPU mode)
	checkCudaErrors(hipSetDevice(devID));

	// * CUDA Capability Information
	//int major = 0, minor = 0;
	//checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, devID));
	//checkCudaErrors(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, devID));
	//printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, _ConvertSMVer2ArchName(major, minor), major, minor);


	/******************************
	* GPU Memory Initialization
	******************************/
	std::cout << "---* Starting GPU Memory Initialization *---\n";
	auto t_init_gpu_1 = std::chrono::high_resolution_clock::now();

	const std::complex<float>* h_data = dataW.data();

	// * Overall cuBlas and cuFFT handle
	CUDAHandle handles(paras.echo_num, paras.range_num);

	// * Overall kernel function configuration
	dim3 block(256);  // block size
	dim3 grid_one_echo((paras.range_num + block.x - 1) / block.x);  // grid size

	// * GPU memory allocation
	hipComplex* d_data = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_data, sizeof(hipComplex) * paras.data_num));
	checkCudaErrors(hipMemcpy(d_data, h_data, sizeof(hipComplex) * paras.data_num, hipMemcpyHostToDevice));  // data (host -> device)

	auto t_init_gpu_2 = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::milliseconds>(t_init_gpu_2 - t_init_gpu_1).count() << "ms\n";
	std::cout << "---* GPU Memory Initialization Over *---\n";
	std::cout << "************************************\n\n";


#ifdef DATA_WRITE_BACK_DATAW
	ioOperation::dataWriteBack(std::string(DIR_PATH) + "dataW.dat", d_data, paras.data_num);
#endif // DATA_WRITE_BACK_DATAW


	/******************************
	* HPC
	******************************/
	if (if_hpc == true) {
		std::cout << "---* Starting HPC *---\n";

		// * Retrieving Velocity Data
		float* h_velocity = new float[paras.echo_num];
		std::transform(dataNOut.cbegin(), dataNOut.cend(), h_velocity, [](const std::vector<float>& v) {return v[1]; });

		float* d_velocity = nullptr;
		checkCudaErrors(hipMalloc((void**)&d_velocity, sizeof(float) * paras.echo_num));
		checkCudaErrors(hipMemcpy(d_velocity, h_velocity, sizeof(float) * paras.echo_num, hipMemcpyHostToDevice));

		auto t_hpc_1 = std::chrono::high_resolution_clock::now();

		// * Starting HPC
		highSpeedCompensation(d_data, d_velocity, paras, handles);

		auto t_hpc_2 = std::chrono::high_resolution_clock::now();

		delete[] h_velocity;
		h_velocity = nullptr;
		checkCudaErrors(hipFree(d_velocity));
		d_velocity = nullptr;

		std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::microseconds>(t_hpc_2 - t_hpc_1).count() << "us\n";
		std::cout << "---* HPC Over *---\n";
		std::cout << "************************************\n\n";
	}


#ifdef DATA_WRITE_BACK_HPC
	ioOperation::dataWriteBack(std::string(DIR_PATH) + "hpc.dat", d_data, paras.data_num);
#endif // DATA_WRITE_BACK_HPC


	/******************
	 * HRRP
	 ******************/
	std::cout << "---* Starting Get HRRP *---\n";

	// * Generate Hamming Window
	float* hamming = nullptr;
	checkCudaErrors(hipMalloc((void**)&hamming, sizeof(float) * paras.range_num));
	genHammingVec << <grid_one_echo, block >> > (hamming, paras.range_num);
	checkCudaErrors(hipDeviceSynchronize());

	// * HRRP - High Resolution Range Profile.
	hipComplex* d_hrrp = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_hrrp, sizeof(hipComplex) * paras.data_num));

	auto t_hrrp_1 = std::chrono::high_resolution_clock::now();

	// d_hrrp = fftshift(fft(hamming .* d_data))
	// d_data = d_data .* repmat(hamming, echo_num, 1)
	getHRRP(d_hrrp, d_data, hamming, paras, handles);

	auto t_hrrp_2 = std::chrono::high_resolution_clock::now();

	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::microseconds>(t_hrrp_2 - t_hrrp_1).count() << "us\n";
	std::cout << "---* Get HRRP Over *---\n";
	std::cout << "************************************\n\n";


#ifdef DATA_WRITE_BACK_HRRP
	ioOperation::dataWriteBack(std::string(DIR_PATH) + "hrrp.dat", d_hrrp, paras.data_num);
#endif // DATA_WRITE_BACK_HRRP


	/******************
	 * Range Alignment and HRRP Centering
	 ******************/
	std::cout << "---* Starting Range Alignment *---\n";
	auto t_ra_1 = std::chrono::high_resolution_clock::now();

	// * Range Alignment
	rangeAlignmentParallel(d_data, hamming, paras, handles);

	auto t_ra_2 = std::chrono::high_resolution_clock::now();

	// * Centering HRRP
	int inter_length = 30;
	HRRPCenter(d_data, inter_length, paras, handles);

#ifdef DATA_WRITE_BACK_RA
	ioOperation::dataWriteBack(std::string(DIR_PATH) + "ra.dat", d_data, paras.data_num);
#endif // DATA_WRITE_BACK_RA

	// * Cutting range profile
	cutRangeProfile(d_data, paras, RANGE_NUM_CUT, handles);

	auto t_ra_3 = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption(aligning)] " << std::chrono::duration_cast<std::chrono::milliseconds>(t_ra_2 - t_ra_1).count() << "ms\n";
	std::cout << "[Time consumption(centering and cutting)] " << std::chrono::duration_cast<std::chrono::milliseconds>(t_ra_3 - t_ra_2).count() << "ms\n";
	std::cout << "---* Range Alignment Over *---\n";
	std::cout << "************************************\n\n";



	/**********************
	 * Phase Compensation
	 * Doppler_Tracking -> RangeVariantPhaseComp -> Fast_Entropy
	 **********************/
	std::cout << "---* Starting Phase Compensation *---\n";
	auto t_pc_1 = std::chrono::high_resolution_clock::now();

	// * Retrieving Azimuth and Pitch Data
	float* h_azimuth = new float[paras.echo_num];
	float* h_pitch = new float[paras.echo_num];
	std::transform(dataNOut.cbegin(), dataNOut.cend(), h_azimuth, [](std::vector<float> v) {return v[2]; });
	std::transform(dataNOut.cbegin(), dataNOut.cend(), h_pitch, [](std::vector<float> v) {return v[3]; });

	// * Range Variant Phase Compensation [todo] optional
	rangeVariantPhaseComp(d_data, h_azimuth, h_pitch, paras, handles);

	delete[] h_azimuth;
	delete[] h_pitch;
	h_pitch = nullptr;
	h_azimuth = nullptr;

	auto t_pc_2 = std::chrono::high_resolution_clock::now();

	// * Fast Entropy
	fastEntropy(d_data, paras.echo_num, paras.range_num, handles);

	auto t_pc_3 = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::microseconds>(t_pc_2 - t_pc_1).count() << "us\n";
	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::microseconds>(t_pc_3 - t_pc_2).count() << "us\n";
	std::cout << "---* Phase Compensation Over *---\n";
	std::cout << "************************************\n\n";


	/**********************
	 * MTRC (Migration Through Range Cell)
	 **********************/
	if (if_mtrc == true) {
		std::cout << "---* Starting MTRC *---\n";
		auto t_mtrc_1 = std::chrono::high_resolution_clock::now();

		auto t_mtrc_2 = std::chrono::high_resolution_clock::now();
		std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::milliseconds>(t_mtrc_2 - t_mtrc_1).count() << "ms\n";
		std::cout << "---* MTRC Over *---\n";
		std::cout << "************************************\n\n";
	}


	/**********************
	* Final Data Write Back
	**********************/
#ifdef DATA_WRITE_BACK_FINAL
	std::cout << "---* Starting Data Write Back *---\n";

	auto t_data_write_back_1 = std::chrono::high_resolution_clock::now();

	ioOperation::dataWriteBack(std::string(DIR_PATH) + "final.dat", d_data, paras.data_num);
	
	auto t_data_write_back_2 = std::chrono::high_resolution_clock::now();
	
	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::milliseconds>(t_data_write_back_2 - t_data_write_back_1).count() << "ms\n";
	std::cout << "---* Data Write Back Over *---\n";
	std::cout << "************************************\n\n";
#endif // DATA_WRITE_BACK_FINAL



	/**********************
	* Free Allocated Memory & Destroy Pointer
	**********************/
	checkCudaErrors(hipFree(d_data));
	checkCudaErrors(hipFree(hamming));
	checkCudaErrors(hipFree(d_hrrp));

	return EXIT_SUCCESS;
}
