#include "hip/hip_runtime.h"
﻿#include "isar_main.cuh"

int ISAR_RD_Imaging_Main_Ku(RadarParameters& paras, const int& datastyle, const vec1D_COM_FLOAT& dataW, const vec2D_FLOAT& dataNOut, const int& optionAligment, const int& optionAPhase, const bool& ifHPC, const bool& ifMTRC) {
	/******************************
	* Init GPU Device
	******************************/
	int devId = findCudaDevice(0, static_cast<const char**>(nullptr));  // initialization GPU Device with no command line arguments
	if (devId == -1) {
		return EXIT_FAILURE;
	}

	
	/******************************
	* Load Echo Data
	******************************/
	std::cout << "---* Starting Load Echo Data *---\n";
	auto tStart_LoadingData = std::chrono::high_resolution_clock::now();

	const std::complex<float>* h_data = dataW.data();
	int num_data = paras.num_echoes * paras.num_range_bins;

	// Overall cuBlas handle
	hipblasHandle_t handle;
	checkCudaErrors(hipblasCreate(&handle));

	//GPU memory mallocation
	hipComplex* d_data = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_data, sizeof(hipComplex) * num_data));
	checkCudaErrors(hipMemcpy(d_data, h_data, sizeof(hipComplex) * num_data, hipMemcpyHostToDevice));  // data (host -> device)
	thrust::device_ptr<comThr> thr_d_data = thrust::device_pointer_cast(reinterpret_cast<comThr*>(d_data));

	auto tEnd_LoadingData = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_LoadingData - tStart_LoadingData).count() << "ms\n";
	std::cout << "---* Load Echo Data Over *---\n";
	std::cout << "************************************\n\n";


	/******************************
	* HPC
	******************************/
	if (ifHPC == true) {
		std::cout << "---* Starting HPC *---\n";
		auto tStart_HPC = std::chrono::high_resolution_clock::now();

		float* h_velocity = new float[paras.num_echoes];
		std::transform(dataNOut.cbegin(), dataNOut.cend(), h_velocity, [](std::vector<float> v) {return v[1]; });

		highSpeedCompensation(d_data, paras.Fs, paras.band_width, paras.Tp, h_velocity, paras.num_echoes, paras.num_range_bins, handle);
		checkCudaErrors(hipDeviceSynchronize());

		delete[] h_velocity;
		h_velocity = nullptr;

		auto tEnd_HPC = std::chrono::high_resolution_clock::now();
		std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_HPC - tStart_HPC).count() << "ms\n";
		std::cout << "---* HPC Over *---\n";
		std::cout << "************************************\n\n";
	}


	/******************
	 * HRRP
	 ******************/
	std::cout << "---* Starting Get HRRP *---\n";
	auto tStart_HRRP = std::chrono::high_resolution_clock::now();

	// fft shift in time domain
	thrust::device_vector<int> fftshift_vec(num_data);
	genFFTShiftVec(fftshift_vec);
	thrust::transform(thrust::device, thr_d_data, thr_d_data + num_data, fftshift_vec.begin(), thr_d_data, \
		[]__host__ __device__(const comThr& x, const int& y) { return x * static_cast<float>(y); });

	getHRRP(d_data, paras.num_echoes, paras.num_range_bins); // HRRP - High Resolution Range Profile

	checkCudaErrors(hipDeviceSynchronize());

	auto tEnd_HRRP = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] " << std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_HRRP - tStart_HRRP).count() << "ms\n";
	std::cout << "---* Get HRRP Over *---\n";
	std::cout << "************************************\n\n";



	/******************
	 * 包络对齐以及距离像序列平移
	 ******************/
	// Range Alignment
	std::cout << "---* Starting Range alignment *---\n";
	auto tStart_RA = std::chrono::high_resolution_clock::now();

	RangeAlignment_linej(d_data, paras, fftshift_vec);

	// HRRPCenter
	unsigned int inter_length = 30;
	HRRPCenter(d_data, paras, inter_length);

	checkCudaErrors(hipDeviceSynchronize());

	auto tEnd_RA = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_RA - tStart_RA).count() << "ms\n";
	std::cout << "---* Range alignment Over *---\n";
	std::cout << "************************************\n\n";


	/******************
	* Cut range profiles
	******************/
	std::cout << "---* Starting Cut range profiles *---\n";
	auto tStart_cut = std::chrono::high_resolution_clock::now();

	const int range_length = 512;

	hipComplex* d_data_cut = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_data_cut, sizeof(hipComplex)* range_length* paras.num_echoes));

	// YSQ 改为求最大值点，确定开始距离单元
	float* range_abs = nullptr;  // 求幅度
	checkCudaErrors(hipMalloc((void**)&range_abs, sizeof(float)* paras.num_range_bins));
	thrust::device_ptr<float> thr_range_abs(range_abs);
	thrust::transform(thrust::device, thr_d_data, thr_d_data + paras.num_range_bins, thr_range_abs,
		[]__host__ __device__(thrust::complex<float> x) { return thrust::abs(x); });
	thrust::device_ptr<float> min_ptr = thrust::max_element(thr_range_abs, thr_range_abs + paras.num_range_bins);

	int mPos = static_cast<int>(&min_ptr[0] - &thr_range_abs[0]);
	paras.Pos = mPos;  // modify paras value

	cutRangeProfile(d_data, d_data_cut, range_length, paras);
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(range_abs));
	auto tEnd_cut = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_cut - tStart_cut).count() << "ms\n";
	std::cout << "---* Cut range profiles Over *---\n";
	std::cout << "************************************\n\n";

	checkCudaErrors(hipFree(d_data));  // use d_data_cut instead
	paras.num_range_bins = range_length;  // modify paras value
	num_data = paras.num_echoes * paras.num_range_bins;


	/**********************
	 * Phase Compensation
	 * 多普勒跟踪 -> 距离向空变的相位补偿 -> 快速最小熵 (Doppler_Tracking -> RangeVariantPhaseComp -> Fast_Entropy)
	 **********************/
	float* h_azimuth = new float[paras.num_echoes];
	float* h_pitch = new float[paras.num_echoes];
	std::transform(dataNOut.cbegin(), dataNOut.cend(), h_azimuth, [](std::vector<float> v) {return v[2]; });  // make zip with range and velocity data
	std::transform(dataNOut.cbegin(), dataNOut.cend(), h_pitch, [](std::vector<float> v) {return v[3]; });

	// Doppler centriod tracing
	std::cout << "---* Starting Doppler centriod tracing *---\n";
	auto tStart_droptrace = std::chrono::high_resolution_clock::now();

	Doppler_Tracking(d_data_cut, paras);
	checkCudaErrors(hipDeviceSynchronize());
	
	auto tEnd_droptrace = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_droptrace - tStart_droptrace).count() << "ms\n";
	std::cout << "---* Doppler centriod tracing Over *---\n";
	std::cout << "************************************\n\n";

	// range variant phase compensation
	std::cout << "---* Starting range variant phase compensation *---\n";
	auto tStart_ran = std::chrono::high_resolution_clock::now();

	RangeVariantPhaseComp(d_data_cut, paras, h_azimuth, h_pitch);

	checkCudaErrors(hipDeviceSynchronize());
	auto tEnd_ran = std::chrono::high_resolution_clock::now();
	std::cout << "[Time consumption] "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(tEnd_ran - tStart_ran).count() << "ms\n";
	std::cout << "---* range variant phase compensation Over *---\n";
	std::cout << "************************************\n\n";

	// 
	Fast_Entropy(d_data_cut, paras);


	// data transfer from GPU to CPU
	std::complex<float>* h_data_cut = new std::complex<float>[num_data];
	checkCudaErrors(hipMemcpy(h_data_cut, d_data_cut, sizeof(hipComplex) * num_data, hipMemcpyDeviceToHost));  // data (device -> host)
	std::vector<std::complex<float>> h_data_cut_vec(h_data_cut, h_data_cut + num_data);

	// wtire to file
	ioOperation io;
	std::string path_out(DIR_PATH);
	path_out.append("test.dat");
	io.WriteFile(path_out, h_data_cut, num_data);
	delete[] h_data_cut;
	h_data_cut = NULL;


	delete[] h_azimuth;
	h_azimuth = nullptr;
	delete[] h_pitch;
	h_pitch = nullptr;
	checkCudaErrors(hipblasDestroy(handle));
	h_data = nullptr;  // h_data cannot be deleted
	return EXIT_SUCCESS;
}