#include "hip/hip_runtime.h"
﻿#include "phase_adjustment.cuh"


void dopplerTracking(hipComplex* d_data_comp, hipComplex* d_phase, hipComplex* d_data, const int& echo_num, const int& range_num, const bool& if_compensation)
{
	int data_num = echo_num * range_num;
	int data_num_less_echo = (echo_num - 1) * range_num;

	dim3 block(DEFAULT_THREAD_PER_BLOCK);
	dim3 grid((data_num + block.x - 1) / block.x);

	// * Applying conjugate multiplication on two successive rows
	hipComplex* d_mul_res = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_mul_res, sizeof(hipComplex) * data_num_less_echo));
	elementwiseMultiplyConjA << <(data_num_less_echo + block.x - 1) / block.x, block >> > (d_data, d_data + range_num, d_mul_res, data_num_less_echo);
	checkCudaErrors(hipDeviceSynchronize());

	// * Sum mulRes in rows
	thrust::device_vector<comThr> xw(echo_num - 1);
	hipComplex* d_xw = reinterpret_cast<hipComplex*>(thrust::raw_pointer_cast(xw.data()));
	sumRows << <echo_num, block, block.x * sizeof(hipComplex) >> > (d_mul_res, d_xw, echo_num - 1, range_num);
	checkCudaErrors(hipDeviceSynchronize());

	// * Getting compensation angle
	thrust::device_vector<float> angle(echo_num - 1);
	thrust::transform(thrust::device, xw.begin(), xw.end(), angle.begin(), \
		[]__host__ __device__(const comThr & x) { return thrust::arg(x); });
	thrust::inclusive_scan(thrust::device, angle.begin(), angle.end(), angle.begin());

	// * Calculating phase using angle
	thrust::device_ptr<comThr> thr_d_phase = thrust::device_pointer_cast(reinterpret_cast<comThr*>(d_phase));
	thr_d_phase[0] = comThr(1.0f, 0.0f);
	thrust::transform(thrust::device, angle.begin(), angle.end(), thr_d_phase + 1, \
		[]__host__ __device__(const float& x) { return thrust::exp(comThr(0.0, -x)); });

	// * Compensation
	if (if_compensation == true) {
		diagMulMat << <grid, block >> > (d_phase, d_data, d_data_comp, range_num, data_num);
		checkCudaErrors(hipDeviceSynchronize());
	}

	// * Free Allocated Space
	checkCudaErrors(hipFree(d_mul_res));
}


__global__ void diagMulMat(hipComplex* d_diag, hipComplex* d_data, hipComplex* d_res, int cols, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < len) {
		d_res[tid] = hipCmulf(d_diag[static_cast<int>(tid / cols)], d_data[tid]);
	}
}


//void rangeVariantPhaseComp(hipComplex* d_data, double* h_azimuth, double* h_pitch, const RadarParameters& paras, const CUDAHandle& handles)
//{
//	// [todo] expanding data width to double
//	// transfer angle information to device, calculating central angle
//	int mid_index = paras.echo_num / 2;
//	float middle_azimuth = h_azimuth[mid_index];
//	float middle_pitch = h_pitch[mid_index];
//
//	float* d_azimuth = nullptr;
//	checkCudaErrors(hipMalloc((void**)&d_azimuth, sizeof(float) * paras.echo_num));
//	checkCudaErrors(hipMemcpy(d_azimuth, h_azimuth, sizeof(float) * paras.echo_num, hipMemcpyHostToDevice));
//
//	float* d_pitch = nullptr;
//	checkCudaErrors(hipMalloc((void**)&d_pitch, sizeof(float) * paras.echo_num));
//	checkCudaErrors(hipMemcpy(d_pitch, h_pitch, sizeof(float) * paras.echo_num, hipMemcpyHostToDevice));
//
//	float resolution = static_cast<float>(LIGHT_SPEED) / static_cast<float>(2 * paras.band_width);
//	float wave_length = static_cast<float>(LIGHT_SPEED) / static_cast<float>(paras.fc);
//
//	// assuming the rotation center is in the middle of the 1D image, generating range vector
//	thrust::device_vector<float> range(paras.range_num);
//	float* d_range = thrust::raw_pointer_cast(range.data());
//	thrust::sequence(thrust::device, range.begin(), range.end(), -float(paras.range_num) / 2.0);
//
//	thrust::transform(thrust::device, range.begin(), range.end(), range.begin(), \
//		[=]__host__ __device__(const float& x) { return x * 2.0f * resolution / wave_length; });
//
//	// calculating turning angle of each echo comparing to that of middle echo signal
//	thrust::device_vector<float> theta(paras.echo_num);
//	float* d_theta = thrust::raw_pointer_cast(theta.data());
//	thrust::device_ptr<float> thr_azimuth = thrust::device_pointer_cast(d_azimuth);
//	thrust::device_ptr<float> thr_pitch = thrust::device_pointer_cast(d_pitch);
//	float mid_x = sinf(middle_pitch / 180 * PI_FLT);
//	float mid_y = cosf(middle_pitch / 180 * PI_FLT) * cosf(middle_azimuth / 180 * PI_FLT);
//	float mid_z = cosf(middle_pitch / 180 * PI_FLT) * sinf(middle_azimuth / 180 * PI_FLT);
//
//	thrust::transform(thrust::device, thr_azimuth, thr_azimuth + paras.echo_num, thr_pitch, theta.begin(), \
//		[=]__host__ __device__(const float& cur_azi, const float& cur_pit)
//	{
//		float x = sinf(cur_pit / 180 * PI_FLT);
//		float y = cosf(cur_pit / 180 * PI_FLT) * cosf(cur_azi / 180 * PI_FLT);
//		float z = cosf(cur_pit / 180 * PI_FLT) * sinf(cur_azi / 180 * PI_FLT);
//
//		float angle = (x * mid_x + y * mid_y + z * mid_z);
//		angle = acosf(angle);
//		float angle2;
//		angle2 = powf(angle, 2.0);
//		return angle2;
//	});
//
//	// build compensation matrix and compensate range sequence
//	float* d_comp_mat = nullptr;
//	checkCudaErrors(hipMalloc((void**)&d_comp_mat, sizeof(float) * paras.data_num));
//	checkCudaErrors(hipMemset(d_comp_mat, 0, sizeof(float) * paras.data_num));
//
//	// 08-05-2020 modified
//	float alpha = 1.0f;
//	checkCudaErrors(hipblasSger(handles.handle, paras.range_num, paras.echo_num, &alpha, d_range, 1, d_theta, 1, d_comp_mat, paras.range_num));
//
//	thrust::device_ptr<float> thr_comp_mat = thrust::device_pointer_cast(d_comp_mat);
//	thrust::device_ptr<comThr> thr_data = thrust::device_pointer_cast(reinterpret_cast<comThr*>(d_data));
//
//	thrust::transform(thrust::device, thr_comp_mat, thr_comp_mat + paras.data_num, thr_data, thr_data, \
//		[]__host__ __device__(const float& x, const comThr & y) { return y * thrust::exp(comThr(0.0, -2 * PI_FLT * x)); });
//
//	// Free Allocated GPU Memory
//	checkCudaErrors(hipFree(d_azimuth));
//	checkCudaErrors(hipFree(d_pitch));
//	checkCudaErrors(hipFree(d_comp_mat));
//}


void fastEntropy(hipComplex*& d_data, const int& echo_num, const int& range_num, const CUDAHandle& handles)
{
	int data_num = echo_num * range_num;

	dim3 block(DEFAULT_THREAD_PER_BLOCK);  // block size
	dim3 grid((data_num + block.x - 1) / block.x);  // grid size

	// * Pre-processing and pre-imaging
	// d_data_abs = abs(d_data)
	float* d_data_abs = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_data_abs, sizeof(float) * data_num));
	elementwiseAbs << <grid, block >> > (d_data, d_data_abs, data_num);
	checkCudaErrors(hipDeviceSynchronize());

	// max_value = max(abs(d_data),[],1);
	float* d_max_val = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_max_val, sizeof(float) * range_num));
	thrust::device_ptr<float> thr_max_value = thrust::device_pointer_cast(d_max_val);
	maxCols << <range_num, block, block.x * sizeof(int) >> > (d_data_abs, d_max_val, echo_num, range_num);
	checkCudaErrors(hipDeviceSynchronize());

	// mean_value = mean(max_value);
	float mean_value = thrust::reduce(thrust::device, thr_max_value, thr_max_value + range_num, 0, thrust::plus<int>()) / static_cast<float>(range_num);
	float threshold = 1.48f * mean_value;

	// tgt_index = find(max_value > mean_value*1.48);
	thrust::device_vector<int> tgt_index(range_num);
	auto end = thrust::copy_if(thrust::make_counting_iterator(0), thrust::make_counting_iterator(range_num), thr_max_value, tgt_index.begin(), \
		[=]__host__ __device__(const float& x) { return (x > threshold); });
	int tgt_num = static_cast<int>(end - tgt_index.begin());
	//tgt_index.resize(tgt_num);

	// tmpData = droptrace(RetData_RA);
	hipComplex* d_data_comp = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_data_comp, sizeof(hipComplex)* data_num));
	hipComplex* d_phase_tmp = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_phase_tmp, sizeof(hipComplex)* echo_num));
	dopplerTracking(d_data_comp, d_phase_tmp, d_data, echo_num, range_num, true);

	if (tgt_num < 5) {
		// * Free allocated GPU memory and return
		checkCudaErrors(hipFree(d_data_abs));
		checkCudaErrors(hipFree(d_max_val));
		checkCudaErrors(hipFree(d_phase_tmp));
		checkCudaErrors(hipFree(d_data));
		d_data = d_data_comp;
		return;
	}

	// d_img = fft(tmpData, [], 1);
	hipComplex* d_img = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_img, sizeof(hipComplex) * data_num));
	thrust::device_ptr<comThr> thr_img = thrust::device_pointer_cast(reinterpret_cast<comThr*>(d_img));
	checkCudaErrors(hipfftExecC2C(handles.plan_all_range_c2c, d_data_comp, d_img, HIPFFT_FORWARD));

	int unit1_num = tgt_num;
	int unit2_num = nextPow2(tgt_num / 2) / 2;

	// sqr_img = abs(image2).^2;
	float* d_sqr_image = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_sqr_image, sizeof(float) * data_num));
	thrust::device_ptr<float> thr_sqr_img = thrust::device_pointer_cast(d_sqr_image);
	thrust::transform(thrust::device, thr_img, thr_img + data_num, thr_sqr_img, \
		[]__host__ __device__(const comThr & x) { return powf(thrust::abs(x), 2); });

	// sqr_image_sum_col = sum(sqr_img);
	float* d_sqr_img_sum_col = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_sqr_img_sum_col, sizeof(float) * range_num));
	thrust::device_ptr<float> thr_sqr_img_sum_col = thrust::device_pointer_cast(d_sqr_img_sum_col);
	sumCols << <range_num, echo_num, echo_num * sizeof(float) >> > (d_sqr_image, d_sqr_img_sum_col, echo_num, range_num);
	checkCudaErrors(hipDeviceSynchronize());

	// * Get entropy
	float* d_mask_entropy = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_mask_entropy, sizeof(float) * data_num));
	thrust::device_ptr<float> thr_mask_entropy = thrust::device_pointer_cast(d_mask_entropy);
	elementwiseDivRep << <grid, block >> > (d_sqr_img_sum_col, d_sqr_image, d_mask_entropy, range_num, data_num);
	checkCudaErrors(hipDeviceSynchronize());

	thrust::transform(thrust::device, thr_mask_entropy, thr_mask_entropy + data_num, thr_mask_entropy, \
		[]__host__ __device__(const float& x) { return -(x * logf(x)); });

	float* d_entropy = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_entropy, sizeof(float) * range_num));
	sumCols << <range_num, echo_num, echo_num * sizeof(float) >> > (d_mask_entropy, d_entropy, echo_num, range_num);
	checkCudaErrors(hipDeviceSynchronize());

	// * Store the index of the first unit1_num largest values of sqr_image_sum_col in select_bin1
	thrust::device_vector<int> idx_sqr_img_sum_col(range_num);
	thrust::sequence(thrust::device, idx_sqr_img_sum_col.begin(), idx_sqr_img_sum_col.end(), 0);
	thrust::stable_sort_by_key(thrust::device, thr_sqr_img_sum_col, thr_sqr_img_sum_col + range_num, idx_sqr_img_sum_col.begin(), thrust::greater<float>());

	thrust::device_vector<int> select_bin1(idx_sqr_img_sum_col.begin(), idx_sqr_img_sum_col.begin() + unit1_num);  // length == unit1_num

	// * Store the index of the first unit2_num smallest values of entropy in select_bin2
	thrust::device_vector<float> select_entropy(unit1_num);
	for (int ii = 0; ii < unit1_num; ii++) {
		select_entropy[ii] = d_entropy[select_bin1[ii]];
	}
	thrust::stable_sort_by_key(thrust::device, select_entropy.begin(), select_entropy.end(), select_bin1.begin());

	thrust::device_vector<int> select_bin2(select_bin1.begin(), select_bin1.begin() + unit2_num);
	int* d_select_bin2 = thrust::raw_pointer_cast(select_bin2.data());

	thrust::sort(thrust::device, select_bin2.begin(), select_bin2.end());

	// * Rebuild echo using select_bin2
	int data_num_unit2 = echo_num * unit2_num;
	float scale_ifft = 1 / static_cast<float>(echo_num);

	hipComplex* d_new_data = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_new_data, sizeof(hipComplex) * data_num_unit2));
	thrust::device_ptr<comThr> thr_new_data = thrust::device_pointer_cast(reinterpret_cast<comThr*>(d_new_data));
	if (unit2_num >= 1024) {
		std::cout << "[fastEntropy/WARN] unit2_num >= 1024, please double-check the data or optimize data layout of selectRangeBins()." << std::endl;
		system("pause");
		exit(EXIT_FAILURE);
	}
	selectRangeBins << <echo_num, unit2_num >> > (d_new_data, d_data, d_select_bin2, echo_num, range_num, unit2_num);
	checkCudaErrors(hipDeviceSynchronize());

	// * Doppler phase tracking
	hipComplex* d_phase_select = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_phase_select, sizeof(hipComplex) * echo_num));
	thrust::device_ptr<comThr> thr_phase_select = thrust::device_pointer_cast(reinterpret_cast<comThr*>(d_phase_select));
	dopplerTracking(d_new_data, d_phase_select, d_new_data, echo_num, unit2_num, false);

	// * Minimum entropy searching
	hipComplex* d_new_data_tmp = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_new_data_tmp, sizeof(hipComplex)* data_num_unit2));
	thrust::device_ptr<comThr> thr_new_data_tmp = thrust::device_pointer_cast(reinterpret_cast<comThr*>(d_new_data_tmp));

	// configure fft handle (along columns)
	hipfftHandle plan_all_unit2_c2c;
	int batch = unit2_num;
	int rank = 1;
	int n[1] = { echo_num };
	int inembed[] = { echo_num };
	int onembed[] = { echo_num };
	int istride = unit2_num;
	int ostride = unit2_num;
	int idist = 1;
	int odist = 1;
	checkCudaErrors(hipfftPlanMany(&plan_all_unit2_c2c, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

	for (int i = 0; i < FAST_ENTROPY_ITERATION_NUM; ++i) {
		diagMulMat << <(unit2_num * echo_num + block.x - 1) / block.x, block >> > (d_phase_select, d_new_data, d_new_data_tmp, unit2_num, echo_num* unit2_num);
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipfftExecC2C(plan_all_unit2_c2c, d_new_data_tmp, d_new_data_tmp, HIPFFT_FORWARD));

		thrust::transform(thrust::device, thr_new_data_tmp, thr_new_data_tmp + data_num_unit2, thr_new_data_tmp, \
			[]__host__ __device__(comThr& x) { return x * thrust::abs(x); });

		checkCudaErrors(hipfftExecC2C(plan_all_unit2_c2c, d_new_data_tmp, d_new_data_tmp, HIPFFT_BACKWARD));
		checkCudaErrors(hipblasCsscal(handles.handle, data_num_unit2, &scale_ifft, d_new_data_tmp, 1));

		thrust::transform(thrust::device, thr_new_data, thr_new_data + data_num_unit2, thr_new_data_tmp, thr_new_data_tmp, \
			[=]__host__ __device__(const comThr& x, const comThr& y) { return thrust::conj(x) * y; });

		sumRows << <echo_num, unit2_num, unit2_num * sizeof(hipComplex) >> > (d_new_data_tmp, d_phase_select, echo_num, unit2_num);
		checkCudaErrors(hipDeviceSynchronize());

		thrust::transform(thrust::device, thr_phase_select, thr_phase_select + echo_num, thr_phase_select, \
			[]__host__ __device__(const comThr& x) { return (x / thrust::abs(x)); });
	}

	diagMulMat << <(data_num + block.x - 1) / block.x, block >> > (d_phase_select, d_data, d_data, range_num, data_num);
	checkCudaErrors(hipDeviceSynchronize());

	// * Free allocated memory
	checkCudaErrors(hipFree(d_data_abs));
	checkCudaErrors(hipFree(d_max_val));
	checkCudaErrors(hipFree(d_data_comp));
	checkCudaErrors(hipFree(d_phase_tmp));
	checkCudaErrors(hipFree(d_img));
	checkCudaErrors(hipFree(d_sqr_image));
	checkCudaErrors(hipFree(d_sqr_img_sum_col));
	checkCudaErrors(hipFree(d_mask_entropy));
	checkCudaErrors(hipFree(d_entropy));
	checkCudaErrors(hipFree(d_new_data));
	checkCudaErrors(hipFree(d_phase_select));
	checkCudaErrors(hipFree(d_new_data_tmp));

	checkCudaErrors(hipfftDestroy(plan_all_unit2_c2c));
}


__global__ void selectRangeBins(hipComplex* d_new_data, hipComplex* d_data, int* select_bin, int echo_num, int range_num, int unit2_num)
{
	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + tid;

	if (idx < echo_num * unit2_num) {
		int row_idx = static_cast<int>(idx / unit2_num);
		int ori_idx = row_idx * range_num + select_bin[tid];

		d_new_data[idx] = d_data[ori_idx];
	}
}
