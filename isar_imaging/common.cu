#include "hip/hip_runtime.h"
﻿#include "common.cuh"


/* CUDAHandle Class */
void CUDAHandle::handleInit(const int& echo_num, const int& range_num)
{
	checkCudaErrors(hipblasCreate(&handle));

	checkCudaErrors(hipfftPlan1d(&plan_all_echo_c2c, range_num, HIPFFT_C2C, echo_num));
	//checkCudaErrors(hipfftPlan1d(&plan_one_echo_c2c, range_num, HIPFFT_C2C, 1));
	//checkCudaErrors(hipfftPlan1d(&plan_one_echo_r2c, range_num, HIPFFT_R2C, 1));
	checkCudaErrors(hipfftPlan1d(&plan_all_echo_r2c, range_num, HIPFFT_R2C, echo_num));
	//checkCudaErrors(hipfftPlan1d(&plan_one_echo_c2r, range_num, HIPFFT_C2R, 1));
	checkCudaErrors(hipfftPlan1d(&plan_all_echo_c2r, range_num, HIPFFT_C2R, echo_num));

	// cuFFT data layout for applying fft to each column along first dimension
	int batch = RANGE_NUM_CUT;
	int rank = 1;
	int n[] = { echo_num };
	int inembed[] = { echo_num };
	int onembed[] = { echo_num };
	int istride = RANGE_NUM_CUT;
	int ostride = RANGE_NUM_CUT;
	int idist = 1;
	int odist = 1;
	checkCudaErrors(hipfftPlanMany(&plan_all_range_c2c, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

	int fft_len = nextPow2(2 * echo_num - 1);
	n[0] = fft_len;
	inembed[0] = fft_len;
	onembed[0] = fft_len;
	checkCudaErrors(hipfftPlanMany(&plan_all_range_c2c_czt, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

	checkCudaErrors(hipfftPlan1d(&plan_all_echo_c2c_cut, RANGE_NUM_CUT, HIPFFT_C2C, echo_num));
}


void CUDAHandle::handleDest()
{
	checkCudaErrors(hipblasDestroy(handle));

	checkCudaErrors(hipfftDestroy(plan_all_echo_c2c));
	//checkCudaErrors(hipfftDestroy(plan_one_echo_c2c));
	//checkCudaErrors(hipfftDestroy(plan_one_echo_r2c));
	checkCudaErrors(hipfftDestroy(plan_all_echo_r2c));
	//checkCudaErrors(hipfftDestroy(plan_one_echo_c2r));
	checkCudaErrors(hipfftDestroy(plan_all_echo_c2r));

	checkCudaErrors(hipfftDestroy(plan_all_range_c2c));
	checkCudaErrors(hipfftDestroy(plan_all_range_c2c_czt));
	checkCudaErrors(hipfftDestroy(plan_all_echo_c2c_cut));
}


void getMax(hipblasHandle_t handle, float* d_vec, int len, int* h_max_idx, float* h_max_val)
{
	checkCudaErrors(hipblasIsamax(handle, len, d_vec, 1, h_max_idx));
	--(*h_max_idx);  // cuBlas using 1-based indexing

	checkCudaErrors(hipMemcpy(h_max_val, d_vec + *h_max_idx, sizeof(float), hipMemcpyDeviceToHost));
}


void getMax(hipblasHandle_t handle, hipComplex* d_vec, int len, int* h_max_idx, hipComplex* h_max_val)
{
	checkCudaErrors(hipblasIcamax(handle, len, d_vec, 1, h_max_idx));
	--(*h_max_idx);  // cuBlas using 1-based indexing

	checkCudaErrors(hipMemcpy(h_max_val, d_vec + *h_max_idx, sizeof(hipComplex), hipMemcpyDeviceToHost));
}


void getMin(hipblasHandle_t handle, float* d_vec, int len, int* min_idx, float* min_val)
{
	checkCudaErrors(hipblasIsamin(handle, len, d_vec, 1, min_idx));
	--(*min_idx);  // cuBlas using 1-based indexing

	checkCudaErrors(hipMemcpy(min_val, d_vec + *min_idx, sizeof(float) * 1, hipMemcpyDeviceToHost));
}


__global__ void elementwiseAbs(hipComplex* a, float* abs, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len) {
		abs[tid] = hipCabsf(a[tid]);
	}
}


__global__ void elementwiseConj(hipComplex* d_data, hipComplex* d_data_conj, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len) {
		d_data_conj[tid] = hipConjf(d_data[tid]);
	}
}


__global__ void elementwiseMultiply(hipComplex* a, hipComplex* b, hipComplex* c, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len) {
		c[tid] = hipCmulf(a[tid], b[tid]);
	}
}


__global__ void elementwiseMultiply(hipDoubleComplex* a, hipDoubleComplex* b, hipDoubleComplex* c, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len) {
		c[tid] = hipCmul(a[tid], b[tid]);
	}
}


__global__ void elementwiseMultiply(float* a, float* b, float* c, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len) {
		c[tid] = a[tid] * b[tid];
	}
}


__global__ void elementwiseMultiply(float* a, hipComplex* b, hipComplex* c, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len) {
		c[tid] = make_hipComplex(a[tid] * b[tid].x, a[tid] * b[tid].y);
	}
}


__global__ void elementwiseMultiplyConjA(hipComplex* a, hipComplex* b, hipComplex* c, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len) {
		c[tid] = hipCmulf(hipConjf(a[tid]), b[tid]);
	}
}


__global__ void elementwiseMultiplyRep(hipComplex* a, hipComplex* b, hipComplex* c, int len_a, int len_b)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len_b) {
		c[tid] = hipCmulf(a[tid % len_a], b[tid]);
	}
}


__global__ void elementwiseMultiplyRep(hipDoubleComplex* a, hipDoubleComplex* b, hipDoubleComplex* c, int len_a, int len_b)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len_b) {
		c[tid] = hipCmul(a[tid % len_a], b[tid]);
	}
}


__global__ void elementwiseMultiplyRep(float* a, hipComplex* b, hipComplex* c, int len_a, int len_b)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len_b) {
		c[tid] = hipCmulf(make_hipComplex(a[tid % len_a], 0.0f), b[tid]);
	}
}


__global__ void elementwiseDiv(float* a, hipComplex* b, hipComplex* c, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len) {
		c[tid] = make_hipComplex(b[tid].x / a[tid], b[tid].y / a[tid]);
	}
}


__global__ void elementwiseDivRep(float* a, float* b, float* c, int len_a, int len_b)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len_b) {
		c[tid] = b[tid] / a[tid % len_a];
	}
}


__global__ void diagMulMat(hipComplex* d_diag, hipComplex* d_data, hipComplex* d_res, int cols, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < len) {
		d_res[tid] = hipCmulf(d_diag[static_cast<int>(tid / cols)], d_data[tid]);
	}
}


__global__ void diagMulMat(float* d_diag, hipComplex* d_data, hipComplex* d_res, int cols, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < len) {
		float tmp = d_diag[static_cast<int>(tid / cols)];
		d_res[tid] = make_hipComplex(tmp * d_data[tid].x, tmp * d_data[tid].y);
	}
}


__global__ void diagMulMat(double* d_diag, double* d_data, double* d_res, int cols, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < len) {
		d_res[tid] = d_diag[static_cast<int>(tid / cols)] * d_data[tid];
	}
}


__global__ void expJ(double* x, hipDoubleComplex* res, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < len) {
		res[tid] = make_hipDoubleComplex(std::cos(x[tid]), std::sin(x[tid]));
	}
}


__global__ void genHammingVec(float* d_hamming, int len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < (len / 2)) {
		int tx = tid;
		d_hamming[tid] = (0.54f - 0.46f * std::cos(2 * PI_FLT * (static_cast<float>(tx) / len - 1)));
	}
	else if (tid < len) {
		int tx = len - tid - 1;
		d_hamming[tid] = (0.54f - 0.46f * std::cos(2 * PI_FLT * (static_cast<float>(tx) / len - 1)));
	}
}


//template <typename T>
//__global__ void getMaxIdx(const T* data, const int dsize, int* result)
//{
//
//	__shared__ volatile T   vals[nTPB];
//	__shared__ volatile int idxs[nTPB];
//	__shared__ volatile int last_block;
//	int idx = threadIdx.x + blockDim.x * blockIdx.x;
//	last_block = 0;
//	T   my_val = FLOAT_MIN;
//	int my_idx = -1;
//	// sweep from global memory
//	while (idx < dsize) {
//		if (data[idx] > my_val) { my_val = data[idx]; my_idx = idx; }
//		idx += blockDim.x * gridDim.x;
//	}
//	// populate shared memory
//	vals[threadIdx.x] = my_val;
//	idxs[threadIdx.x] = my_idx;
//	__syncthreads();
//	// sweep in shared memory
//	for (int i = (nTPB >> 1); i > 0; i >>= 1) {
//		if (threadIdx.x < i)
//			if (vals[threadIdx.x] < vals[threadIdx.x + i]) { vals[threadIdx.x] = vals[threadIdx.x + i]; idxs[threadIdx.x] = idxs[threadIdx.x + i]; }
//		__syncthreads();
//	}
//	// perform block-level reduction
//	if (!threadIdx.x) {
//		blk_vals[blockIdx.x] = vals[0];
//		blk_idxs[blockIdx.x] = idxs[0];
//		if (atomicAdd(&blk_num, 1) == gridDim.x - 1) // then I am the last block
//			last_block = 1;
//	}
//	__syncthreads();
//	if (last_block) {
//		idx = threadIdx.x;
//		my_val = FLOAT_MIN;
//		my_idx = -1;
//		while (idx < gridDim.x) {
//			if (blk_vals[idx] > my_val) { my_val = blk_vals[idx]; my_idx = blk_idxs[idx]; }
//			idx += blockDim.x;
//		}
//		// populate shared memory
//		vals[threadIdx.x] = my_val;
//		idxs[threadIdx.x] = my_idx;
//		__syncthreads();
//		// sweep in shared memory
//		for (int i = (nTPB >> 1); i > 0; i >>= 1) {
//			if (threadIdx.x < i)
//				if (vals[threadIdx.x] < vals[threadIdx.x + i]) { vals[threadIdx.x] = vals[threadIdx.x + i]; idxs[threadIdx.x] = idxs[threadIdx.x + i]; }
//			__syncthreads();
//		}
//		if (!threadIdx.x)
//			*result = idxs[0];
//	}
//}


__global__ void maxCols(float* d_data, float* d_max_clos, int rows, int cols)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int nTPB = blockDim.x;

	// [todo] Possible optimization:  halve the number of threads and size of shared memory assigned for each block.
	// Perform a reduction within the block to compute the final maximum value.
	// sdata_max_cols_int store the index of the maximum value in each block.
	extern __shared__ int sdata_max_cols_int[];
	sdata_max_cols_int[tid] = tid * cols + bid;
	__syncthreads();

	for (int s = (nTPB >> 1); s > 0; s >>= 1) {
		if (tid < s) {
			if (d_data[sdata_max_cols_int[tid]] < d_data[sdata_max_cols_int[tid + s]]) {
				sdata_max_cols_int[tid] = sdata_max_cols_int[tid + s];
			}
		}
		__syncthreads();
	}

	if (tid == 0) {
		d_max_clos[bid] = d_data[sdata_max_cols_int[0]];
	}
}


__global__ void sumCols(float* d_data, float* d_sum_clos, int rows, int cols)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int nTPB = blockDim.x;

	// [todo] Possible optimization:  halve the number of threads and size of shared memory assigned for each block.
	// Perform a reduction within the block to compute the final sum
	extern __shared__ float sdata_sum_cols_flt[];
	sdata_sum_cols_flt[tid] = d_data[tid * cols + bid];
	__syncthreads();

	for (int s = (nTPB >> 1); s > 0; s >>= 1) {
		if (tid < s) {
			sdata_sum_cols_flt[tid] += sdata_sum_cols_flt[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) {
		d_sum_clos[bid] = sdata_sum_cols_flt[0];
	}
}


__global__ void sumRows(hipComplex* d_data, hipComplex* d_sum_rows, int rows, int cols)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;

	hipComplex t_sum = make_hipComplex(0.0f, 0.0f);
	for (int i = tid; i < cols; i += blockDim.x) {
		t_sum = hipCaddf(t_sum, d_data[bid * cols + i]);
	}

	// Perform a reduction within the block to compute the final sum
	extern __shared__ hipComplex sdata_sum_rows_com_flt[];
	sdata_sum_rows_com_flt[tid] = t_sum;
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata_sum_rows_com_flt[tid] = hipCaddf(sdata_sum_rows_com_flt[tid], sdata_sum_rows_com_flt[tid + s]);
		}
		__syncthreads();
	}

	if (tid == 0) {
		d_sum_rows[bid] = sdata_sum_rows_com_flt[0];
	}
}


__global__ void sumRows(float* d_data, float* d_sum_rows, int rows, int cols)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;

	float t_sum = 0.0f;
	for (int i = tid; i < cols; i += blockDim.x) {
		t_sum = t_sum + d_data[bid * cols + i];
	}

	// Perform a reduction within the block to compute the final sum
	extern __shared__ float sdata_sum_rows_flt[];
	sdata_sum_rows_flt[tid] = t_sum;
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata_sum_rows_flt[tid] = sdata_sum_rows_flt[tid] + sdata_sum_rows_flt[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) {
		d_sum_rows[bid] = sdata_sum_rows_flt[0];
	}
}


void cutRangeProfile(hipComplex* d_data_cut, hipComplex* d_data, RadarParameters& paras, const int& range_num_cut, const CUDAHandle& handles)
{
	int data_num_cut = paras.echo_num * range_num_cut;

	dim3 block(DEFAULT_THREAD_PER_BLOCK);  // block size
	dim3 grid((data_num_cut + block.x - 1) / block.x);  // grid size

	// max(abs(d_data(1,:)))
	int range_abs_max_idx = 0;
	hipComplex range_abs_max_val = make_hipComplex(0.0f, 0.0f);
	getMax(handles.handle, d_data, paras.range_num, &range_abs_max_idx, &range_abs_max_val);

	int offset_l = range_abs_max_idx - range_num_cut / 2;
	int offset_r = range_abs_max_idx + range_num_cut / 2;
	if (offset_l < 0 || offset_r >= paras.range_num) {
		std::cout << "[cutRangeProfile/WARN] Invalid range_num_cut! Probably too long.\n" << std::endl;
		system("pause");
		exit(EXIT_FAILURE);
	}

	cutRangeProfileHelper << <grid, block >> > (d_data, d_data_cut, data_num_cut, offset_l, range_num_cut, paras.range_num);
	checkCudaErrors(hipDeviceSynchronize());

	// updating values of paras
	paras.range_num = range_num_cut;
	paras.data_num = paras.echo_num * paras.range_num;
}


__global__ void cutRangeProfileHelper(hipComplex* d_in, hipComplex* d_out, int data_num_cut, int offset, int range_num_cut, int range_num)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < data_num_cut) {
		d_out[tid] = d_in[(tid / range_num_cut) * range_num + offset + tid % range_num_cut];
	}
}


int nextPow2(int N) {
	int n = 1;
	while (N >> 1) {
		n = n << 1;
		N = N >> 1;
	}
	n = n << 1;
	return n;
}


__global__ void setNumInArray(int* d_data, int* d_index, int val, int d_index_len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < d_index_len) {
		d_data[d_index[tid]] = val;
	}
}


void getHRRP(hipComplex* d_hrrp, hipComplex* d_data, const RadarParameters& paras, const CUDAHandle& handles)
{
	dim3 block(DEFAULT_THREAD_PER_BLOCK);  // block size

	// fft
	checkCudaErrors(hipfftExecC2C(handles.plan_all_echo_c2c, d_data, d_hrrp, HIPFFT_FORWARD));
	// fftshift
	ifftshiftRows << <dim3(((paras.range_num / 2) + block.x - 1) / block.x, paras.echo_num), block >> > (d_hrrp, paras.range_num);
	checkCudaErrors(hipDeviceSynchronize());
}


float getTurnAngle(const float& azimuth1, const float& pitching1, const float& azimuth2, const float& pitching2) {
	vec1D_FLT vec_1({ std::sin(pitching1 / 180 * PI_FLT), \
		std::cos(pitching1 / 180 * PI_FLT) * std::cos(azimuth1 / 180 * PI_FLT), \
		std::cos(pitching1 / 180 * PI_FLT) * std::sin(azimuth1 / 180 * PI_FLT) });

	vec1D_FLT vec_2({ std::sin(pitching2 / 180 * PI_FLT), \
		std::cos(pitching2 / 180 * PI_FLT) * std::cos(azimuth2 / 180 * PI_FLT), \
		std::cos(pitching2 / 180 * PI_FLT) * std::sin(azimuth2 / 180 * PI_FLT) });

	float ret = std::acos(vec_1[0] * vec_2[0] + vec_1[1] * vec_2[1] + vec_1[2] * vec_2[2]) / PI_FLT * 180;

	return ret;
}

int turnAngleLine(vec1D_FLT* turnAngle, const vec1D_FLT& azimuth, const vec1D_FLT& pitching) {

	vec1D_INT idx;
	int pitchingSize = static_cast<int>(pitching.size());
	for (int i = 0; i < pitchingSize - 1; ++i) {
		if (std::abs(pitching[i + 1] - pitching[i]) > 0.2) {
			idx.push_back(i);
		}
	}

	vec1D_INT blkBeginNum;
	vec1D_INT blkEndNum;
	vec1D_INT blkLen;
	int idxSize = idx.empty() ? 1 : static_cast<int>(idx.size());

	blkBeginNum.insert(blkBeginNum.cend(), -1);
	blkBeginNum.insert(blkBeginNum.cend(), idx.begin(), idx.end());
	int blkSize = static_cast<int>(blkBeginNum.size());
	std::for_each(blkBeginNum.begin(), blkBeginNum.end(), [](int& x) {x++; });

	blkEndNum.insert(blkEndNum.cend(), idx.begin(), idx.end());
	blkEndNum.insert(blkEndNum.cend(), pitchingSize - 1);

	blkLen.assign(blkSize, 0);
	std::transform(blkEndNum.cbegin(), blkEndNum.cend(), blkBeginNum.cbegin(), blkLen.begin(), [](const int& end, const int& begin) {return end - begin + 1; });

	turnAngle->assign(pitchingSize, 0);
	for (int blkIdx = 0; blkIdx < blkSize; ++blkIdx) {
		int N = blkLen[blkIdx];
		int stride = (N < 21) ? 1 : 20;
		for (int i = stride; i < N; i += stride) {
			int currentPulseNum = blkBeginNum[blkIdx] + i;
			float azimuth1 = azimuth[currentPulseNum - stride];
			float azimuth2 = azimuth[currentPulseNum];
			float pitching1 = pitching[currentPulseNum - stride];
			float pitching2 = pitching[currentPulseNum];
			float turnAngleSingle = getTurnAngle(azimuth1, pitching1, azimuth2, pitching2);
			turnAngle->at(currentPulseNum) = turnAngle->at(currentPulseNum - stride) + turnAngleSingle;  // angle superposition
		}
		int turnAngleSize = static_cast<int>(turnAngle->size());
		for (int i = 0; i < turnAngleSize; ++i) {
			turnAngle->at(i) = std::abs(turnAngle->at(i));
		}
		if (N >= 21) {
			vec1D_INT x = [=]() {
				vec1D_INT v;
				for (int i = 0; (i + stride) <= N; i += stride) {
					v.push_back(i);
				}
				return v;
			}();  // todo: range generate
			vec1D_FLT Y = [=]() {
				vec1D_FLT v;
				int xSize = static_cast<int>(x.size());
				for (int i = 0; i < xSize; ++i) {  // interpolation movement
					v.push_back(turnAngle->at(x[i]));
				}
				return v;
			}();
			vec1D_FLT turnAngleInterp = [=]() {
				vec1D_FLT v;
				for (int i = 0; i < N; ++i) {
					v.push_back(interpolate(x, Y, i, false));
				}
				return v;
			}();
			turnAngle->erase(turnAngle->cbegin() + blkBeginNum[blkIdx], turnAngle->cbegin() + blkEndNum[blkIdx] + 1);
			turnAngle->insert(turnAngle->cbegin() + blkBeginNum[blkIdx], turnAngleInterp.cbegin(), turnAngleInterp.cend());
		}

		if (blkIdx > 0) {
			for (int i = blkBeginNum[blkIdx]; i <= blkEndNum[blkIdx]; ++i) {
				turnAngle->at(i) += turnAngle->at(blkEndNum[blkIdx - 1]);
			}
		}
	}

	return EXIT_SUCCESS;
}


float interpolate(const vec1D_INT& xData, const vec1D_FLT& yData, const int& x, const bool& extrapolate) {
	int size = static_cast<int>(xData.size());

	int i = 0;  // find left end of interval for interpolation
	if (x >= xData[size - 2]) {  // special case: beyond right end
		i = size - 2;
	}
	else {
		while (x > xData[i + 1]) i++;
	}
	float xL = static_cast<float>(xData[i]);
	float yL = yData[i];
	float xR = static_cast<float>(xData[i + 1]);
	float yR = yData[i + 1];  // points on either side (unless beyond ends)
	if (!extrapolate) {  // if beyond ends of array and not extrapolating
		if (x < xL) yR = yL;
		if (x > xR) yL = yR;
	}

	float dydx = (yR - yL) / (xR - xL);  // gradient

	return yL + dydx * (x - xL);  // linear interpolation
}


int uniformSampling(vec1D_INT* dataWFileSn, vec2D_DBL* dataNOut, vec1D_FLT* turnAngleOut, \
	const vec2D_DBL& dataN, const vec1D_FLT& turnAngle, const int& sampling_stride, const int& window_head, const int& window_len)
{
	// dataWFileSn = window_head:sampling_stride:window_end;
	for (int i = 0; i < window_len; ++i) {
		dataWFileSn->at(i) = window_head + i * sampling_stride;
	}

	// DataNOut = DataN(window_head:sampling_stride:window_end, : );
	std::transform(dataWFileSn->cbegin(), dataWFileSn->cend(), dataNOut->begin(), [&](const int& x) {return dataN[x]; });

	// TurnAngleOut = abs(TurnAngle(window_head:sampling_stride:window_end));
	std::transform(dataWFileSn->cbegin(), dataWFileSn->cend(), turnAngleOut->begin(), [&](const int& x) {return std::abs(turnAngle[x]); });

	return EXIT_SUCCESS;
}


int nonUniformSampling() {
	return EXIT_SUCCESS;
}


/* ioOperation Class */
void ioOperation::ioInit(std::string* INTERMEDIATE_DIR, const std::string& file_path, const int& polar_type, const int& data_type)
{
	m_file_path = file_path;
	m_polar_type = polar_type;
	m_data_type = data_type;

	// validating file_path
	fs::path fs_file_path(m_file_path);
	if (fs::is_regular_file(fs_file_path) == false) {
		std::cout << "[ioInit/WARN] Invalid file path!\n";
		return;
	}
	m_dir_path = fs_file_path.parent_path().string();

	// assign global variables
	*INTERMEDIATE_DIR = m_dir_path + std::string("\\intermediate\\");
}


int ioOperation::getSystemParas(RadarParameters* paras, int* frame_len, int* frame_num)
{
	std::ifstream ifs;
	ifs.open(m_file_path, std::ios_base::in | std::ios_base::binary);
	if (!ifs) {
		std::cout << "[getSystemParas/WARN] Cannot open file " << m_file_path << " !\n";
		return EXIT_FAILURE;
	}

	ifs.seekg(0, ifs.beg);

	uint32_t temp[36]{};
	ifs.read((char*)&temp, sizeof(uint32_t) * 36);  // 144 bytes in total

	// [Caution] Possibly bits overflow
	*frame_len = static_cast<int>(temp[4] * 4);  // length of frame, including frame head and orthogonal demodulation data.(unit: Byte)
	paras->fc = static_cast<long long>(temp[12] * 1e6);  // signal carrier frequency
	paras->band_width = static_cast<long long>(temp[13] * 1e6);  // signal band width
	paras->Tp = static_cast<double>(temp[15] / 1e6);  // pulse width
	paras->Fs = static_cast<int>((temp[17] % static_cast<int>(std::pow(2, 16))) * 1e6);  // sampling frequency
	*frame_num = static_cast<int>(fs::file_size(fs::path(m_file_path))) / *frame_len;  // total frame number in file

	ifs.close();

	return EXIT_SUCCESS;
}


int ioOperation::readKuIFDSALLNBStretch(vec2D_DBL* dataN, vec1D_INT* stretchIndex, vec1D_FLT* turnAngle, \
	const RadarParameters& paras, const int& frame_len, const int& frame_num)
{
	std::ifstream ifs;
	ifs.open(m_file_path, std::ios_base::in | std::ios_base::binary);
	if (!ifs) {
		std::cout << "[readKuIFDSALLNBStretch/WARN] Cannot open file " << m_file_path << " !\n";
		return EXIT_FAILURE;
	}

	dataN->resize(frame_num);
	stretchIndex->resize(frame_num);

	vec1D_FLT azimuthVec(frame_num);  // todo: expanding to double?
	vec1D_FLT pitchingVec(frame_num);

	//uint64_t sysTime = 0;
	uint32_t headerData[11]{};

	double range = 0;  // unit: m
	double velocity = 0;  // unit: m/s
	double azimuth = 0;
	double pitching = 0;

	//float timeYear = 0;  // only need to be read once
	//float timeMonth = 0;
	//float timeDay = 0;
	for (int i = 0; i < frame_num; i++) {
		stretchIndex->at(i) = i * frame_len + 256;

		//ifs.seekg(i * frame_len + 40, ifs.beg);
		ifs.seekg(i * frame_len + 48, ifs.beg);

		//ifs.read((char*)&sysTime, sizeof(uint64_t));

		ifs.read((char*)&headerData, sizeof(uint32_t) * 11);

		range = static_cast<double>(headerData[7]) * 0.1;
		velocity = static_cast<double>(headerData[8]);
		azimuth = static_cast<double>(headerData[9]);
		pitching = static_cast<double>(headerData[10]);

		// [caution]: possible bit overflow
		velocity = (velocity - (velocity > std::pow(2, 31) ? std::pow(2, 32) : 0)) * 0.1;

		azimuth = (azimuth - (azimuth > std::pow(2, 31) ? std::pow(2, 32) : 0)) * (360.0 / std::pow(2, 24));
		azimuth += (azimuth < 0 ? 360.0 : 0);

		pitching = (pitching - (pitching > std::pow(2, 31) ? std::pow(2, 32) : 0)) * (360.0 / std::pow(2, 24));
		pitching += (pitching < 0 ? 360.0 : 0);

		//ifs.seekg(i * frame_len + 32, ifs.beg);
		//if (i == 0) {
		//	ifs.read((char*)&timeYear, sizeof(uint16_t));
		//	ifs.read((char*)&timeMonth, sizeof(uint8_t));
		//	ifs.read((char*)&timeDay, sizeof(uint8_t));
		//}

		//dataN->at(i) = vec1D_DBL({ range, velocity, azimuth, pitching, static_cast<double>(sysTime), static_cast<double>(timeYear), static_cast<double>(timeMonth), static_cast<double>(timeDay) });
		dataN->at(i) = vec1D_DBL({ range, velocity, azimuth, pitching });
		azimuthVec[i] = static_cast<float>(azimuth);
		pitchingVec[i] = static_cast<float>(pitching);
	}

	turnAngleLine(turnAngle, azimuthVec, pitchingVec);

	ifs.close();

	return EXIT_SUCCESS;
}


int ioOperation::getKuDataStretch(vec1D_COM_FLT* dataW, vec1D_INT* frameHeader, \
	const vec1D_INT& stretchIndex, const int& frame_len, const vec1D_INT& dataWFileSn, const int& window_len)
{
	std::ifstream ifs;
	ifs.open(m_file_path, std::ios_base::in | std::ios_base::binary);
	if (!ifs) {
		std::cout << "[getKuDataStretch/WARN] Cannot open file " << m_file_path << " !\n";
		return EXIT_FAILURE;
	}

	int dataADTempSize = (frame_len - 256) / 2;
	int16_t* dataADTemp = new int16_t[dataADTempSize];
	
	for (int i = 0; i < window_len; ++i) {
		//fseek(fid1, StretchIndex(DataW_FileSn(i), 1), 'bof');
		ifs.seekg(stretchIndex[dataWFileSn[i]], ifs.beg);

		//DataAD = fread(fid1, (StretchIndex(DataW_FileSn(i), 2) - 256) / 2, 'int16');
		ifs.read((char*)dataADTemp, dataADTempSize * sizeof(int16_t));

		//data_AD = DataAD(1:2 : end) + 1i * DataAD(2:2 : end);
		//DataW(i, :) = data_AD.';
		for (int j = 0; (j + 1) < dataADTempSize; j += 2) {
			dataW->at(i * (dataADTempSize / 2) + (j / 2)) = std::complex<float>(static_cast<float>(dataADTemp[j]), static_cast<float>(dataADTemp[j + 1]));
		}
	}
	delete[] dataADTemp;
	dataADTemp = nullptr;

	/*
	fseek(fid1, StretchIndex(DataW_FileSn(1), 1) - 256, 'bof');
	DataRead = fread(fid1, 108, 'uint8');
	FrameHeader = [DataRead(1:12, 1); DataRead(101:104, 1); DataRead(77:92, 1); DataRead(97:100, 1); DataRead(33:38, 1); DataRead(31, 1); DataRead(105:108, 1); DataRead(61:64, 1); ];
	*/
	ifs.seekg(stretchIndex[dataWFileSn[0]] - 256, ifs.beg);

	uint8_t frameHeaderTemp[108]{};
	ifs.read((char*)&frameHeaderTemp, sizeof(frameHeaderTemp));

	frameHeader->insert(frameHeader->cend(), frameHeaderTemp + 0, frameHeaderTemp + 12);
	frameHeader->insert(frameHeader->cend(), frameHeaderTemp + 100, frameHeaderTemp + 104);
	frameHeader->insert(frameHeader->cend(), frameHeaderTemp + 76, frameHeaderTemp + 92);
	frameHeader->insert(frameHeader->cend(), frameHeaderTemp + 96, frameHeaderTemp + 100);
	frameHeader->insert(frameHeader->cend(), frameHeaderTemp + 32, frameHeaderTemp + 38);
	frameHeader->insert(frameHeader->cend(), frameHeaderTemp + 30, frameHeaderTemp + 31);
	frameHeader->insert(frameHeader->cend(), frameHeaderTemp + 104, frameHeaderTemp + 108);
	frameHeader->insert(frameHeader->cend(), frameHeaderTemp + 60, frameHeaderTemp + 64);

	ifs.close();

	return EXIT_SUCCESS;
}


int ioOperation::writeFile(const std::string& outFilePath, const hipComplex* data, const  size_t& data_size)
{
	std::ofstream ofs(outFilePath);
	if (!ofs.is_open()) {
		std::cout << "[writeFile/WARN] Cannot open the file: " << outFilePath << std::endl;
		return EXIT_FAILURE;
	}

	for (int idx = 0; idx < data_size; idx++) {
		ofs << std::fixed << std::setprecision(5) << data[idx].x << "\n" << data[idx].y << "\n";
	}

	ofs.close();
	return EXIT_SUCCESS;
}


int ioOperation::writeFile(const std::string& outFilePath, const hipDoubleComplex* data, const  size_t& data_size)
{
	std::ofstream ofs(outFilePath);
	if (!ofs.is_open()) {
		std::cout << "[writeFile/WARN] Cannot open the file: " << outFilePath << std::endl;
		return EXIT_FAILURE;
	}

	for (int idx = 0; idx < data_size; idx++) {
		ofs << std::fixed << std::setprecision(5) << data[idx].x << "\n" << data[idx].y << "\n";
	}

	ofs.close();
	return EXIT_SUCCESS;
}


int ioOperation::writeFile(const std::string& outFilePath, const float* data, const  size_t& data_size)
{
	std::ofstream ofs(outFilePath);
	if (!ofs.is_open()) {
		std::cout << "[writeFile/WARN] Cannot open the file: " << outFilePath << std::endl;
		return EXIT_FAILURE;
	}

	for (int idx = 0; idx < data_size; idx++) {
		ofs << std::fixed << std::setprecision(5) << data[idx] << "\n";
	}

	ofs.close();
	return EXIT_SUCCESS;
}


int ioOperation::writeFile(const std::string& outFilePath, const double* data, const  size_t& data_size)
{
	std::ofstream ofs(outFilePath);
	if (!ofs.is_open()) {
		std::cout << "[writeFile/WARN] Cannot open the file: " << outFilePath << std::endl;
		return EXIT_FAILURE;
	}

	for (int idx = 0; idx < data_size; idx++) {
		ofs << std::fixed << std::setprecision(5) << data[idx] << "\n";
	}

	ofs.close();
	return EXIT_SUCCESS;
}
